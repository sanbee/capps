#include "hip/hip_runtime.h"
// -*- C -*-
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <Utils.h>
#include <math.h>
#include <cuUtils.h>
/* #include <cuWTerm.h> */
/* #include <thrust/host_vector.h> */
/* #include <thrust/device_vector.h> */

#define USE_AUTO 
//#undef USE_AUTO 
#define BLOCKSIZE 128
#define GRIDSIZE (2048*2048/128)

namespace casa{
  //
  //--------------------------------------------
  //
  void * allocateDeviceBuffer(int N)
  {
    void *d_buf;
    hipMalloc((void**)&d_buf, N);
    if (hipGetLastError() != hipSuccess)
      {
	fprintf(stderr, "Cuda error: Failed to allocate\n");
	return 0;
      }
    
    return d_buf;
  }
  //
  //--------------------------------------------
  //
  int sendBufferToDevice(void *d_buf, void *h_buf, int N)
  {
    hipMemcpy(d_buf, h_buf, N, hipMemcpyHostToDevice);
    if (hipGetLastError() != hipSuccess)
      {
	fprintf(stderr, "Cuda error: Failed to send\n");
	return 0;
      }
    return 1;
  }
  //
  //--------------------------------------------
  //
  int getBufferFromDevice(void *h_buf, void *d_buf, int N)
  {
    hipMemcpy(h_buf, d_buf, N, hipMemcpyDeviceToHost);
    if (hipGetLastError() != hipSuccess)
      {
	fprintf(stderr, "Cuda error: Failed to get\n");
	return 0;
      }
    return 1;
  }
  //
  //--------------------------------------------
  //
  int makeCUFFTPlan(hipfftHandle *plan, int NX, int NY, hipfftType type /*HIPFFT_C2C*/)
  {
    /* Create a 2D FFT plan. */
    if (hipfftPlan2d(plan, NX, NY, type) != HIPFFT_SUCCESS)
      {
	fprintf(stderr, "CUFFT Error: Unable to create plan\n");
	return 0;
      }
    return 1;
  }
  //
  //--------------------------------------------
  //
  int setCompatibilityMode(hipfftHandle& plan, cufftCompatibility mode /*CUFFT_COMPATIBILITY_NATIVE*/)
  {
    if (cufftSetCompatibilityMode(plan, mode)!= HIPFFT_SUCCESS)
      {
	fprintf(stderr, "CUFFT Error: Unable to set compatibility mode to native\n");
	return 0;
      }
    return 1;
  }
  //
  //--------------------------------------------
  //
  int inPlaceCUFFTC2C(hipfftHandle& plan, hipfftComplex *d_buf, int dir /*HIPFFT_FORWARD */)
  {
    if (hipfftExecC2C(plan, d_buf, d_buf, dir) != HIPFFT_SUCCESS)
      {
	fprintf(stderr, "CUFFT Error: Unable to execute plan\n");
	return 0;
      }
    hipDeviceSynchronize();
    return 1;
  }
  //
  //--------------------------------------------
  //
  hipError_t
    freeHost(void* h_mem, memoryMode memMode)
  {
    if( PINNED == memMode ) {
      return hipHostFree(h_mem);
    }
    else {
      free(h_mem);
    }
    return hipSuccess;
  }
  //
  //--------------------------------------------
  //
  hipError_t
    mallocHost(void** h_mem ,uint memSize, memoryMode memMode, bool wc)
  {
    if( PINNED == memMode ) {
#if CUDART_VERSION >= 2020
      return hipHostAlloc( h_mem, memSize, (wc) ? hipHostMallocWriteCombined : 0 );
#else
      if (wc) {printf("Write-Combined unavailable on CUDART_VERSION less than 2020, running is: %d", CUDART_VERSION);
        return hipHostMalloc( h_mem, memSize );
#endif
      }
      else { // PAGEABLE memory mode
        *h_mem = malloc( memSize );
      }
      
      return hipSuccess;
    }
    //
    //--------------------------------------------
    //
    hipError_t
      memCpy(void* sink, void* source, uint memSize, hipMemcpyKind direction, memoryMode memMode)
    {
      /* if( PINNED == memMode ) { */
      /*   return hipMemcpyAsync( sink, source, memSize, direction, 0); */
      /* } */
      /* else { */
      /*   return hipMemcpy( sink, source, memSize, direction); */
      /* } */
      return hipMemcpy( sink, source, memSize, direction);
    }
    //
    //--------------------------------------------
    //
    void cpu_wTermApplySky(hipfftComplex* screen, const int nx, const int ny,
			   const int tileWidthX, const int tileWidthY, 
			   const double wPixel,  const float sampling, 
			   const double wScale,  const int inner,      
			   const bool isNoOp)
    {
      double wValue=(wPixel*wPixel)/wScale;
      double twoPiW=2.0*M_PI*double(wValue);
      int convSize = nx;
      
      if (!isNoOp)
      	{
      	  for (int iy=-inner/2;iy<inner/2;iy++)
      	    {
      	      double m=sampling*double(iy);
      	      double msq=m*m;
      	      for (int ix=-inner/2;ix<inner/2;ix++)
      		{
      		  double l=sampling*double(ix);
      		  double rsq=l*l+msq;
      		  if(rsq<1.0)
      		    {
      		      double phase=twoPiW*(sqrt(1.0-rsq)-1.0);
		      hipfftComplex w;w.x=cos(phase); w.y=sin(phase);
		      screen[ix+convSize/2 + (iy+convSize/2)*ny]=
			hipCmulf(screen[ix+convSize/2 + (iy+convSize/2)*ny], w); 
      		      /* float wre=cos(phase), wim=sin(phase); */
      		      /* float re=screen[ix+convSize/2 + (iy+convSize/2)*ny].x, */
      		      /* 	im=screen[ix+convSize/2 + (iy+convSize/2)*ny].y; */
      		      /* screen[ix+convSize/2 + (iy+convSize/2)*ny].x=re*wre - im*wim; */
      		      /* screen[ix+convSize/2 + (iy+convSize/2)*ny].y=re*wim + im*wre; */
      		    }
      		}
      	    }
      	}
    }
    //
    //===========================================
    //--------------------------------------------
    //
    __global__ void kernel_wTermApplySky(hipfftComplex* screen, 
					 const hipfftComplex* aTerm, 
					 const int nx, const int ny,
					 const int tileWidthX, const int tileWidthY, 
					 const double wPixel,
					 const float sampling, const double wScale, 
					 const int inner,      const bool isNoOp)
    {
      unsigned int col = tileWidthX*blockIdx.x + threadIdx.x ;
      unsigned int row = tileWidthY*blockIdx.y + threadIdx.y ;
      int originx=nx/2, originy=ny/2, tix, tiy;
      int ix=row-inner/2, iy=col-inner/2;
      tix=ix+originx; tiy=iy+originy;
      
      
      double m=sampling*double(ix), l=sampling*double(iy);
      double rsq=(l*l+m*m);

      if (rsq<1.0)
	{
	  double wValue=(wPixel*wPixel)/wScale;
	  double phase=2.0*M_PI*double(wValue)*(sqrt(1.0-rsq)-1.0);
	  hipfftComplex w; __sincosf(phase, &(w.y),&(w.x));
	  screen[tix*ny+tiy] = hipCmulf(w,aTerm[tix*ny+tiy]);
	  //screen[tix*ny+tiy] = w;
	}
      else
	{
	  screen[tix*ny+tiy] = make_hipFloatComplex(0.0,0.0);
	}
    }
    //
    //--------------------------------------------
    //
    void wTermApplySky(hipfftComplex* screen,  
		       const hipfftComplex* aTerm,  
		       const int& nx, const int& ny,
		       const int tileWidthX, const int tileWidthY, 
		       const double& wPixel,
		       const float& sampling, const double& wScale, 
		       const int& inner,      const bool& isNoOp)
    {
#ifdef USE_AUTO
      {
	int WIDTH=ny;
	dim3 dimGrid ( WIDTH/tileWidthX , WIDTH/tileWidthY ,1 ) ;
	dim3 dimBlock( tileWidthX, tileWidthY, 1 ) ;
	
	kernel_wTermApplySky <<<dimGrid,dimBlock>>> (screen, aTerm, nx, ny, tileWidthX, tileWidthY,wPixel, sampling,
						     wScale, inner,isNoOp);
      }
#else
      {
	dim3 dimGrid ( GRIDSIZE , 1 ,1 ) ;
	dim3 dimBlock( BLOCKSIZE,1,1);
	kernel_wTermApplySky <<<dimGrid,dimBlock>>> (screen, aTerm, nx, ny, tileWidthX, tileWidthY,wPixel, sampling, 
						     wScale, inner,isNoOp);
      }
#endif
    }
    //
    //===========================================
    //--------------------------------------------
    //
    __global__ void kernel_setBuf(hipfftComplex *d_buf, const int nx, const int ny, 
				  const int tileWidthX, const int tileWidthY, 
				  hipfftComplex val)
    {
      int WIDTH=ny;
      
      // calculate thread id
      unsigned int col = tileWidthX*blockIdx.x + threadIdx.x ;
      unsigned int row = tileWidthY*blockIdx.y + threadIdx.y ;
      d_buf[row*WIDTH+col] = val;
    }
    //
    //--------------------------------------------
    //
    void setBuf(hipfftComplex *d_buf, const int nx, const int ny, 
		const int tileWidthX, const int tileWidthY, 
		hipfftComplex val)
    {
#ifdef USE_AUTO
      {
	int WIDTH=ny;
	dim3 dimGrid ( WIDTH/tileWidthX , WIDTH/tileWidthY ,1 ) ;
	dim3 dimBlock( tileWidthX, tileWidthY, 1 ) ;
	
	kernel_setBuf<<<dimGrid,dimBlock>>> ( d_buf,nx,ny,tileWidthX, tileWidthY,val);
      }
#else
      {
	dim3 dimGrid ( GRIDSIZE ,1 ,1 ) ;
	dim3 dimBlock( BLOCKSIZE, 1, 1 ) ;
	
	kernel_setBuf<<<dimGrid,dimBlock>>> ( d_buf,nx,ny,tileWidthX, tileWidthY,val);
      }
#endif
    }
    //
    //===========================================
    //--------------------------------------------
    //
    __global__ void kernel_mulBuf(hipfftComplex *target_d_buf, const hipfftComplex* source_d_buf, 
				  const int nx, const int ny, const int tileWidthX, const int tileWidthY)
    {
      int WIDTH=ny;
      
      // calculate thread id
      unsigned int col = tileWidthX*blockIdx.x + threadIdx.x ;
      unsigned int row = tileWidthY*blockIdx.y + threadIdx.y ;
      target_d_buf[row*WIDTH+col] = hipCmulf(target_d_buf[row*WIDTH+col], source_d_buf[row*WIDTH+col]);
    }
    //
    //--------------------------------------------
    //
    void mulBuf(hipfftComplex *target_d_buf, const hipfftComplex* source_d_buf, 
		const int& nx, const int& ny, const int tileWidthX, const int tileWidthY)
    {
#ifdef USE_AUTO
      {
	int WIDTH=ny;
	dim3 dimGrid ( WIDTH/tileWidthX , WIDTH/tileWidthY ,1 ) ;
	dim3 dimBlock( tileWidthX, tileWidthY, 1 ) ;
	
	kernel_mulBuf<<<dimGrid,dimBlock>>>(target_d_buf, source_d_buf, nx,ny,tileWidthX, tileWidthY);
      }
#else
      {
	dim3 dimGrid ( GRIDSIZE, 1 ,1 ) ;
	dim3 dimBlock( BLOCKSIZE, 1, 1 ) ;
	
	kernel_mulBuf<<<dimGrid,dimBlock>>>(target_d_buf, source_d_buf, nx,ny,tileWidthX, tileWidthY);
      }
#endif
    }
    //
    //--------------------------------------------
    //
    void cpuflip(hipfftComplex *buf, const int nx, const int ny, const int tileWidthX, const int tileWidthY)
    {
      int cx=nx/2, cy=ny/2;
      //      hipComplex sign=make_hipFloatComplex(1.0,0.0);
      
      for (int i=0; i<cx; i++)
	for (int j=0; j< cy; j++)
	  {
	    hipfftComplex tmp;
	    // if (((cx+i)+(cy+j))%2 == 0) sign.x = -1.0; else sign.x = 1.0;
	    //tmp=hipCmulf(buf[i+j*ny],sign);
	    tmp=buf[i+j*ny];

	    //if (((i)+(j))%2 == 0) sign.x = -1.0;  else sign.x = 1.0;
	    //buf[i+j*ny] = hipCmulf(buf[cx+i + (cy+j)*ny],sign);
	    buf[i+j*ny] = buf[cx+i + (cy+j)*ny];

	    buf[cx+i + (cy+j)*ny] = tmp;
	  }
      for (int i=cx; i < nx; i++)
	for (int j=0; j < cy; j++)
	  {
	    hipfftComplex tmp;

	    /* if (((i-cx)+(j+cy))%2 == 0) sign.x = -1.0; else sign.x = 1.0; */
	    /* tmp=hipCmulf(buf[i-cx +(j+cy)*ny],sign); */
	    tmp=buf[i-cx +(j+cy)*ny];

	    /* if (((i)+(j))%2 == 0) sign.x = -1.0; else sign.x = 1.0; */
	    /* buf[i-cx +(j+cy)*ny] = hipCmulf(buf[i + j*ny],sign); */
	    buf[i-cx +(j+cy)*ny] = buf[i + j*ny];

	    buf[i + j*ny] = tmp;
	  }
    }
    //
    //===========================================
    // Following is the GPU kernel equivalent of the cpuflip function
    //
    //  +--------------------+
    //  |         :          |
    //  |         :          |
    //  |    1    :     2    |
    //  |         :          |
    //  |....................|
    //  |         :          |
    //  |    4    :     3    |
    //  |         :          |
    //  |         :          |
    //  +--------------------+
    //
    // This function copies data from quadrant 1 to 3 (and from 3 to 1) and 4 to 2 (and from 2 to 4).
    // While copying, it also now flips the sign of the pixel values if the target pixel (i,j) satisfies (i+j)%2 != 0.
    // This therefore effectively combines the flipSign() kernel in flip() kernel itself (i.e., flipSign() does not
    // need to be envoked).  This saves ~15% in run-time.
    //
#define FLIPSIGN(i,j,ny,val) ({if (((i)+(j))%2 != 0) {(val[(i)+(j)*(ny)]).x *=-1.0; (val[(i)+(j)*(ny)]).y *= -1.0;}})
    __global__ void kernel_flip(hipfftComplex *buf, const int nx, const int ny, const int tileWidthX, const int tileWidthY)
    {
      // calculate thread id
      unsigned int i = tileWidthX*blockIdx.x + threadIdx.x ;
      unsigned int j = tileWidthY*blockIdx.y + threadIdx.y ;
      unsigned int cx=nx/2, cy=ny/2;
      hipComplex sign=make_hipFloatComplex(1.0,0.0);

      hipfftComplex tmp;

      if (i < cx)
	{
	  if (((cx+i)+(cy+j))%2 == 0) sign.x=1.0; else sign.x = -1.0;
	  tmp=hipCmulf(buf[i+j*ny],sign);
	  
	  if ((i+j)%2 == 0) sign.x=1.0; else sign.x = -1.0;
	  buf[i+j*ny] = hipCmulf(buf[cx+i + (cy+j)*ny],sign);

	  buf[cx+i + (cy+j)*ny] = tmp;

	  /* The commented out code below is cleaner code, but which
	     runs ~10% slower!!!  Don't understand why. */

	  /* tmp=buf[i+j*ny]; */
	  /* buf[i+j*ny] = buf[cx+i + (cy+j)*ny]; */
	  /* FLIPSIGN(i,j,ny,buf); */
	  /* tx=cx+i; ty=cy+j; */
	  /* buf[tx + ty*ny] = tmp; */
	  /* FLIPSIGN(tx, ty,ny, buf); */
	}
      else
	{
	  if ((i+j)%2 == 0) sign.x=1.0; else sign.x = -1.0;
	  tmp=hipCmulf(buf[i-cx +(j+cy)*ny],sign);

	  if (((i-cx)+(j+cy))%2 == 0) sign.x=1.0; else sign.x = -1.0;
	  buf[i-cx +(j+cy)*ny] = hipCmulf(buf[i + j*ny],sign);
	    
	  buf[i + j*ny] = tmp;

	  /* The commented out code below is cleaner code, but which
	     runs ~10% slower!!!  Don't understand why. */

	  /* unsigned int tx,ty; */
	  /* tx=i-cx; ty=j+cy; */
	  /* tmp=buf[tx +ty*ny]; */
	  /* buf[i-cx +(j+cy)*ny] = buf[i + j*ny]; */
	  /* FLIPSIGN(tx, ty,ny, buf); */
	  /* buf[i + j*ny] = tmp; */
	  /* FLIPSIGN(i,j,ny,buf); */
	}
    }
    //
    //--------------------------------------------
    //
    void flip(hipfftComplex *buf, const int nx, const int ny, const int tileWidthX, const int tileWidthY)
    {
#ifdef USE_AUTO
      {
	dim3 dimGrid ( nx/tileWidthX , ny/(2*tileWidthY) ,1 ) ;
	dim3 dimBlock( tileWidthX, tileWidthY, 1 ) ;
	
	kernel_flip<<<dimGrid,dimBlock>>>(buf, nx,ny,tileWidthX, tileWidthY);
      }
#else
      {
	dim3 dimGrid ( GRIDSIZE , 1 ,1 ) ;
	dim3 dimBlock( BLOCKSIZE, 1, 1 ) ;
	
	kernel_flip<<<dimGrid,dimBlock>>>(buf, nx,ny,tileWidthX, tileWidthY);
      }
#endif
    }
    //
    //============================================
    //--------------------------------------------
    //
    __global__ void kernel_flipSign(hipfftComplex *buf, const int nx, const int ny, const int tileWidthX, const int tileWidthY)
    {
      // calculate thread id
      unsigned int i = tileWidthX*blockIdx.x + threadIdx.x ;
      unsigned int j = tileWidthY*blockIdx.y + threadIdx.y ;
      float sign;
      {
	//	sign=__pow(-1.0,(float)(i+j));
	if ((i+j)%2 == 0) sign=1.0; else sign=-1.0;
	buf[i + j*ny].x = buf[i + j*ny].x*sign;
	buf[i + j*ny].y = buf[i + j*ny].y*sign;
      }
    }
    //
    //--------------------------------------------
    //
    void flipSign(hipfftComplex *buf, const int nx, const int ny, const int tileWidthX, const int tileWidthY)
    {
#ifdef USE_AUTO
      {
	dim3 dimGrid ( nx/tileWidthX , ny/tileWidthY ,1 ) ;
	dim3 dimBlock( tileWidthY, tileWidthY, 1 ) ;
	kernel_flipSign<<<dimGrid,dimBlock>>>(buf, nx,ny,tileWidthX, tileWidthY);
      }
#else
      {
	dim3 dimGrid ( GRIDSIZE , 1 ,1 ) ;
	dim3 dimBlock( BLOCKSIZE, 1, 1 ) ;
	kernel_flipSign<<<dimGrid,dimBlock>>>(buf, nx,ny,tileWidthX, tileWidthY);
      }
#endif
    }

  };
