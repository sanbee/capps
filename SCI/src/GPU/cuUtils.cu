// -*- C -*-
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <Utils.h>

namespace casa{
  void * allocateDeviceBuffer(int N)
  {
    void *d_buf;
    hipMalloc((void**)&d_buf, N);
    if (hipGetLastError() != hipSuccess)
      {
	fprintf(stderr, "Cuda error: Failed to allocate\n");
	return 0;
      }
    
    return d_buf;
  }
  
  int sendBufferToDevice(void *d_buf, void *h_buf, int N)
  {
    hipMemcpy(d_buf, h_buf, N, hipMemcpyHostToDevice);
    if (hipGetLastError() != hipSuccess)
      {
	fprintf(stderr, "Cuda error: Failed to send\n");
	return 0;
      }
    return 1;
  }
  
  int getBufferFromDevice(void *h_buf, void *d_buf, int N)
  {
    hipMemcpy(h_buf, d_buf, N, hipMemcpyDeviceToHost);
    if (hipGetLastError() != hipSuccess)
      {
	fprintf(stderr, "Cuda error: Failed to get\n");
	return 0;
      }
    return 1;
    
  }
  
  int makeCUFFTPlan(hipfftHandle *plan, int NX, int NY, hipfftType type /*HIPFFT_C2C*/)
  {
    /* Create a 2D FFT plan. */
    if (hipfftPlan2d(plan, NX, NY, type) != HIPFFT_SUCCESS)
      {
	fprintf(stderr, "CUFFT Error: Unable to create plan\n");
	return 0;
      }
    return 1;
  }
  
  int setCompatibilityMode(hipfftHandle& plan, cufftCompatibility mode /*CUFFT_COMPATIBILITY_NATIVE*/)
  {
    if (cufftSetCompatibilityMode(plan, mode)!= HIPFFT_SUCCESS)
      {
	fprintf(stderr, "CUFFT Error: Unable to set compatibility mode to native\n");
	return 0;
      }
    return 1;
  }
  
  int inPlaceCUFFTC2C(hipfftHandle& plan, hipfftComplex *d_buf, int dir /*HIPFFT_FORWARD */)
  {
    if (hipfftExecC2C(plan, d_buf, d_buf, dir) != HIPFFT_SUCCESS)
      {
	fprintf(stderr, "CUFFT Error: Unable to execute plan\n");
	return 0;
      }
    hipDeviceSynchronize();
    return 1;
  }

  hipError_t
    freeHost(void* h_mem, memoryMode memMode)
  {
    if( PINNED == memMode ) {
      return hipHostFree(h_mem);
    }
    else {
      free(h_mem);
    }
    return hipSuccess;
  }
  
  
  hipError_t
    mallocHost(void** h_mem ,uint memSize, memoryMode memMode, bool wc)
  {
    if( PINNED == memMode ) {
#if CUDART_VERSION >= 2020
      return hipHostAlloc( h_mem, memSize, (wc) ? hipHostMallocWriteCombined : 0 );
#else
      if (wc) {printf("Write-Combined unavailable on CUDART_VERSION less than 2020, running is: %d", CUDART_VERSION);
        return hipHostMalloc( h_mem, memSize );
#endif
      }
      else { // PAGEABLE memory mode
        *h_mem = malloc( memSize );
      }
      
      return hipSuccess;
    }
    
    hipError_t
      memCpy(void* sink, void* source, uint memSize, hipMemcpyKind direction, memoryMode memMode)
    {
      /* if( PINNED == memMode ) { */
      /*   return hipMemcpyAsync( sink, source, memSize, direction, 0); */
      /* } */
      /* else { */
      /*   return hipMemcpy( sink, source, memSize, direction); */
      /* } */
        return hipMemcpy( sink, source, memSize, direction);
    }
    
  };
