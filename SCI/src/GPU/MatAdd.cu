#include "hip/hip_runtime.h"
#include<stdio.h>

//#include<cutil_inline.h>



#define BLOCK_SIZE 16



__global__ static void AddKernel(float *d_Buff1, float *d_Buff2, float *d_Buff3, size_t pitch, int iMatSizeM, int iMatSizeN)
  
{
  
  const int tidx = blockDim.x * blockIdx.x + threadIdx.x;
  
  const int tidy = blockDim.y * blockIdx.y + threadIdx.y;
  
  
  
  int index = pitch/sizeof(float);
  
  
  
  if(tidx<iMatSizeM && tidy<iMatSizeN)
    
    {
      
      d_Buff3[tidx * index + tidy] = d_Buff1[tidx * index + tidy] + d_Buff2[tidx * index + tidy];
      
    }
  
}



void printMatrix(float *pflMat, int iMatSizeM, int iMatSizeN)
  
{
  
  for(int idxM = 0; idxM < iMatSizeM; idxM++)
    
    {
      
      for(int idxN = 0; idxN < iMatSizeN; idxN++)
	
	{
	  
	  printf("%f\t",pflMat[(idxM * iMatSizeN) + idxN]);
	  
	}
      
      printf("\n");
      
    }
  
  printf("\n");
  
}

int main()
  
{
  
  int iMatSizeM=0,iMatSizeN=0;
  
  printf("Enter size of Matrix(M*N):");
  
  scanf("%d %d",&iMatSizeM,&iMatSizeN);
  
  
  
  float *h_flMat1 = (float*)malloc(iMatSizeM * iMatSizeN * sizeof(float));
  
  float *h_flMat2 = (float*)malloc(iMatSizeM * iMatSizeN * sizeof(float));
  
  float *h_flMatSum = (float*)malloc(iMatSizeM * iMatSizeN * sizeof(float));
  
  
  
  for(int j=0;j<(iMatSizeM*iMatSizeN);j++)
    
    {
      
      h_flMat1[j]=(float)rand()/(float)RAND_MAX;
      
      h_flMat2[j]=(float)rand()/(float)RAND_MAX;
      
    }
  
  
  
  printf("Matrix 1\n");
  
  printMatrix(h_flMat1, iMatSizeM, iMatSizeN);
  
  printf("Matrix 2\n");
  
  printMatrix(h_flMat2, iMatSizeM, iMatSizeN);
  
  
  
  float *d_flMat1, *d_flMat2, *d_flMatSum;;
  
  size_t d_MatPitch;
  
  hipMallocPitch((void**)&d_flMat1,&d_MatPitch,iMatSizeN*sizeof(float),iMatSizeM);
  
  hipMallocPitch((void**)&d_flMat2,&d_MatPitch,iMatSizeN*sizeof(float),iMatSizeM);
  
  hipMallocPitch((void**)&d_flMatSum,&d_MatPitch,iMatSizeN*sizeof(float),iMatSizeM);
  
  
  
  hipMemcpy2D(d_flMat1,d_MatPitch,h_flMat1,iMatSizeN * sizeof(float), iMatSizeN * sizeof(float), iMatSizeM, hipMemcpyHostToDevice);
  
  hipMemcpy2D(d_flMat2,d_MatPitch,h_flMat2,iMatSizeN * sizeof(float), iMatSizeN * sizeof(float), iMatSizeM, hipMemcpyHostToDevice);
  
  
  
  dim3 blocks(1,1,1);
  
  dim3 threadsperblock(BLOCK_SIZE,BLOCK_SIZE,1);
  
  blocks.x=((iMatSizeM/BLOCK_SIZE) + (((iMatSizeM)%BLOCK_SIZE)==0?0:1));
  
  blocks.y=((iMatSizeN/BLOCK_SIZE) + (((iMatSizeN)%BLOCK_SIZE)==0?0:1));
  
  
  
  AddKernel<<<blocks, threadsperblock>>>(d_flMat1, d_flMat2, d_flMatSum, d_MatPitch, iMatSizeM,iMatSizeN);
  
  
  
  hipDeviceSynchronize();
  
  
  
  hipMemcpy2D(h_flMatSum, iMatSizeN * sizeof(float),d_flMatSum, d_MatPitch, iMatSizeN * sizeof(float), iMatSizeM, hipMemcpyDeviceToHost);
  
  
  
  hipFree(d_flMat1);
  
  hipFree(d_flMat2);
  
  hipFree(d_flMatSum);
  
  
  
  printf("Matrix Sum\n");
  
  printMatrix(h_flMatSum, iMatSizeM, iMatSizeN);
  
}
