#include "hip/hip_runtime.h"
// -*- C++ -*-
//#include <synthesis/TransformMachines/cDataToGridImpl.h>
#include "cDataToGridImpl.h"
#include <typeinfo>
#include <stdio.h>

namespace casa{

template <class T>
__global__ void kernel_cuDataToGridImpl_p(T* gridStore,  Int* gridShape /*4-elements*/,
					  VBStore* vbs, Matrix<Double> *sumwt,
					  const Bool dopsf,
					  const Int* polMap_ptr, const Int *chanMap_ptr,
					  const Double *uvwScale_ptr, const Double *offset_ptr,
					  const Double *dphase_ptr, Int XThGrid=0, Int YThGrid=0
					  )
{
  printf("SubGrid: %d %d\n", blockIdx.x, blockIdx.y);
};

template <class T>
void cuDataToGridImpl_p(T* gridStore,  Int* gridShape /*4-elements*/,
					  VBStore* vbs, Matrix<Double> *sumwt,
					  const Bool dopsf,
					  const Int* polMap_ptr, const Int *chanMap_ptr,
					  const Double *uvwScale_ptr, const Double *offset_ptr,
					  const Double *dphase_ptr, Int XThGrid=0, Int YThGrid=0
					  )
{
  Int NB=10, NT=5;
  dim3 dimBlock ( NB, NB, 1 ) ;
  dim3 dimThread( NT, NT, 1 ) ;

  kernel_cuDataToGridImpl_p<<<dimBlock,dimThread>>>(gridStore, gridShape, vbs, sumwt, dopsf, polMap_ptr, chanMap_ptr,
						    uvwScale_ptr, offset_ptr, dphase_ptr, XThGrid, YThGrid);
  hipError_t err=hipGetLastError();
  if (err != hipSuccess)
    {
      cerr << "###Cuda error: Failed to run the kernel " << hipGetErrorString (err) << endl;
      exit(0);
    }
};

template <class T>
void cDataToGridImpl_p(T* gridStore,  Int* gridShape /*4-elements*/,
		       VBStore* vbs, Matrix<Double>* sumwt,
		       const Bool dopsf,
		       const Int* polMap_ptr, const Int *chanMap_ptr,
		       const Double *uvwScale_ptr, const Double *offset_ptr,
		       const Double *dphase_ptr, Int XThGrid=0, Int YThGrid=0)
{
  //LogIO log_l(LogOrigin("ProtoVR[R&D]","DataToGridImpl_p"));

  // Complex tmp;
  // Bool isGridSinglePrecision=(typeid(gridStore[0]) == typeid(tmp));
  // cerr << "cuisGridSinglePrecision = " << isGridSinglePrecision << endl;


  Int nGridPol, nGridChan, nx, ny, nw, nCFFreq;
  Int targetIMChan, targetIMPol, rbeg, rend;
  Int startChan, endChan;
  Bool accumCFs;

  Float sampling[2],scaledSampling[2];
  Int support[2],loc[3], iloc[4],tiloc[4],scaledSupport[2];
  Int convOrigin[4], gridInc_l[4], cfInc_l[4]; 
  Double pos[2], off[3];
  Int igrdpos[4];
  
  Complex phasor, nvalue, wt;
  Complex norm;
  Int cfShape[4];
  Bool Dummy;
  Bool * flagCube_ptr=vbs->flagCube_p.getStorage(Dummy);
  Bool * rowFlag_ptr = vbs->rowFlag_p.getStorage(Dummy);
  Float * imgWts_ptr = vbs->imagingWeight_p.getStorage(Dummy);
  Complex * visCube_ptr = vbs->visCube_p.getStorage(Dummy);
  Double *sumWt_ptr=sumwt->getStorage(Dummy);

  //  Vector<Double> pointingOffset(cfb.getPointingOffset());
  // Double *pointingOffset_ptr=vbs->cfBSt_p.pointingOffset,
  //   *cached_PointingOffset_ptr=cached_PointingOffset_p.getStorage(Dummy);
  Double *pointingOffset_ptr=vbs->cfBSt_p.pointingOffset,
    *cached_PointingOffset_ptr=NULL;

  Int vbSpw=vbs->spwID_p;
    

  rbeg = vbs->beginRow_p;
  rend = vbs->endRow_p;
  
  nx=gridShape[0]; ny=gridShape[1];
  nGridPol=gridShape[2]; nGridChan=gridShape[3];
  Bool gDummy;
  
  Double *freq=vbs->freq_p.getStorage(Dummy);
  
  cacheAxisIncrements(gridShape, gridInc_l);

  nCFFreq = vbs->cfBSt_p.shape[0]; // shape[0]: nChan, shape[1]: nW, shape[2]: nPol
  nw = vbs->cfBSt_p.shape[1];

  iloc[0]=iloc[1]=iloc[2]=iloc[3]=0;
  Int nDataChan=vbs->nDataChan_p,
    nDataPol = vbs->nDataPol_p;
  accumCFs=vbs->accumCFs_p;
  if (accumCFs)
    {
      startChan = vbs->startChan_p;
      endChan = vbs->endChan_p;
    }
  else 
    {
      startChan = 0;
      endChan = vbs->nDataChan_p;
    }


  //  cerr << "ProtoVR: " << rbeg << " " << rend << " " << startChan << " " << endChan << " " << nDataChan << " " << nDataPol << endl;
  
  // Bool finitePointingOffsets= (
  // 			       (fabs(pointingOffset_ptr[0])>0) ||  
  // 			       (fabs(pointingOffset_ptr[1])>0)
  // 			       );
  Bool finitePointingOffsets = False;
  for(Int irow=rbeg; irow< rend; irow++)
    {   
      if(!(*(rowFlag_ptr+irow)))
	{   
	  for(Int ichan=startChan; ichan< endChan; ichan++)
	    {
	      if (*(imgWts_ptr + ichan+irow*nDataChan)!=0.0) 
		{  
		  targetIMChan=chanMap_ptr[ichan];
		  
		  if((targetIMChan>=0) && (targetIMChan<nGridChan)) 
		    {
		      Double dataWVal = vbs->vb_p->uvw()(irow)(2);
		      
		      Int wndx = (int)(sqrt(vbs->cfBSt_p.wIncr*abs(dataWVal*freq[ichan]/C::c)));
		      
		      Int cfFreqNdx;
		      if (vbs->conjBeams_p) cfFreqNdx = vbs->cfBSt_p.conjFreqNdxMap[vbSpw][ichan];
		      else cfFreqNdx = vbs->cfBSt_p.freqNdxMap[vbSpw][ichan];
		      
		      Float s;
		      s=vbs->cfBSt_p.CFBStorage->sampling;
		      support[0]=vbs->cfBSt_p.CFBStorage->xSupport;
		      support[1]=vbs->cfBSt_p.CFBStorage->ySupport;
		      
		      sampling[0] = sampling[1] = SynthesisUtils::nint(s);
		      
		      const Double *uvw_ptr=vbs->uvw_p.getStorage(Dummy);
			// *uvwScale_ptr=uvwScale_p.getStorage(Dummy),
			// *offset_ptr=offset_p.getStorage(Dummy);;
		      
		      csgrid(pos,loc,off, phasor, irow, vbs->uvw_p, dphase_ptr[irow], freq[ichan], 
			    uvwScale_ptr, offset_ptr, sampling);
		      
		      Float cfblc[2], cftrc[2];
		      //		    pos[0]=1024.1;pos[1]=1025.6;
		      Bool onMyGrid=
			ccomputeSupport(vbs, XThGrid, YThGrid, support, sampling, pos, loc,cfblc,cftrc);
		      // 		    if (onGrid(nx, ny, nw, loc, support)) 
		      if (onMyGrid)
			{
			  
			  Int iblc[2], itrc[2];
			  
			  iblc[0]=SynthesisUtils::nint((cfblc[0]-pos[0]));///sampling[0]);
			  iblc[1]=SynthesisUtils::nint((cfblc[1]-pos[1]));///sampling[1]);
			  itrc[0]=SynthesisUtils::nint((cftrc[0]-pos[0]));///sampling[0]);
			  itrc[1]=SynthesisUtils::nint((cftrc[1]-pos[1]));///sampling[1]); 
			  
			  Int dx=abs(itrc[0]-iblc[0])+1, dy=abs(itrc[1]-iblc[1])+1;
			  Float cfFractioanlArea = (dx*dy)/(float)square(abs(support[1]+support[0])+1);
			  
			  // if ((irow < 3) && (ichan == 2))
			  //   {
			  //     // cerr << irow << "#[" << XThGrid << "," << YThGrid << "] " 
			  //     // 	 << (cfblc[0]) << " " << (cftrc[0]) << " " 
			  //     // 	 << (cfblc[1]) << " " << (cftrc[1]) << " " 
			  //     // 	 << dx << " " << dy << " " << (dx*dy) << " " << cfFractionalArea
			  //     // 	 << endl;
			  //     // cerr << irow << "#[" << XThGrid << "," << YThGrid << "] " 
			  //     // 	 << (pos[0]) << " " << (pos[1]) << endl;
			      
			  //     cerr << irow << " [" << XThGrid << "," << YThGrid << "] " 
			  // 	   << iblc[0] << " " << itrc[0] << " " 
			  // 	   << iblc[1] << " " << itrc[1] << " " << rend
			  // 	   << endl;
			  //   }
			  
			  // Loop over all image-plane polarization planes.
			  for(Int ipol=0; ipol< nDataPol; ipol++) 
			    { 
			      if((!(*(flagCube_ptr + ipol + ichan*nDataPol + irow*nDataPol*nDataChan))))
				{  
				  targetIMPol=polMap_ptr[ipol];
				  if ((targetIMPol>=0) && (targetIMPol<nGridPol)) 
				    {
				      igrdpos[2]=targetIMPol; igrdpos[3]=targetIMChan;
				      
				      // if(accumCFs)     allPolNChanDone_l(ipol,ichan,irow)=True;
				      if(dopsf) nvalue=Complex(*(imgWts_ptr + ichan + irow*nDataChan));
				      else      nvalue= *(imgWts_ptr+ichan+irow*nDataChan)*
						  (*(visCube_ptr+ipol+ichan*nDataPol+irow*nDataChan*nDataPol)*phasor);
				      
				      norm = 0.0;
				      // for (uInt mRow=0;mRow<conjMNdx[ipol].nelements(); mRow++) 
				      // for (uInt mRow=0;mRow<vbs.cfBSt_p.conjMuellerElementsIndex[ipol].nelements(); mRow++) 
				      Bool foundCFPeak=False;
				      for (uInt mRow=0;mRow<vbs->cfBSt_p.nMueller; mRow++) 
					{
					  Complex* convFuncV;
					  // CUWORK:  Essentially CFC.getCellPtr(FNDX, WNDX, POLNDX)
					  // CUWORK: CFC wrapper
					  convFuncV=cgetConvFunc_p(cfShape, vbs, dataWVal, cfFreqNdx, wndx, 
								  vbs->cfBSt_p.muellerElementsIndex,
								  vbs->cfBSt_p.conjMuellerElementsIndex, ipol,  mRow);
					  
					  convOrigin[0]=cfShape[0]/2;
					  convOrigin[1]=cfShape[1]/2;
					  convOrigin[2]=cfShape[2]/2;
					  convOrigin[3]=cfShape[3]/2;
					  Bool psfOnly=((dopsf==True) && (accumCFs==False));
					  // // CUWORK: Convert to a global function with native types

					  // Int cachedPhaseGradNX=cached_phaseGrad_p.shape()[0],
					  //   cachedPhaseGradNY=cached_phaseGrad_p.shape()[1];
					  // Complex *cached_PhaseGrad_ptr=cached_phaseGrad_p.getStorage(Dummy);
					  Int cachedPhaseGradNX=0,cachedPhaseGradNY=0;
					  Complex *cached_PhaseGrad_ptr=NULL;
					  
					  if (finitePointingOffsets && !psfOnly)
					    ccachePhaseGrad_g(cached_PhaseGrad_ptr, cachedPhaseGradNX, cachedPhaseGradNY,	
							     cached_PointingOffset_ptr, pointingOffset_ptr, cfShape, convOrigin);//, cfRefFreq);//, vbs->imRefFreq());
					  
					  cacheAxisIncrements(cfShape, cfInc_l);
					  //cerr << gridShape[0] << " " << gridShape[1] << " " << gridInc_p[0] << " " << gridInc_p[0] << endl;
					  norm += caccumulateOnGrid(gridStore, gridInc_l, cached_PhaseGrad_ptr, 
								   cachedPhaseGradNX, cachedPhaseGradNY,
								   convFuncV, cfInc_l, nvalue,dataWVal,
								   iblc,itrc,/*support,*/ sampling, off, 
								   convOrigin, cfShape, loc, igrdpos,
								   finitePointingOffsets,psfOnly,foundCFPeak);
					}
				      
				      //sumwt(targetIMPol,targetIMChan) += vbs->imagingWeight_p(ichan, irow);//*abs(norm);
				      //cerr << sumwt << " " << targetIMPol << " " << targetIMChan << " " << vbs->imagingWeight_p(ichan, irow) << " " << abs(norm) << endl;
				      // Int dx=abs(itrc[0]-iblc[0]+1), dy=abs(itrc[1]-iblc[1]+1);
				      // Float cfPixArea = (float)square(abs(support[0]-support[1]+1));
				      
				      // Accumulate data weight only when the CF peak was used. This can also be done via
				      // fractional area of the CF used, and probably should be done that for high accuracy 
				      // using the norm of the CF.
				      //if (foundCFPeak) 
				      *(sumWt_ptr+targetIMPol+targetIMChan*nGridPol)+= *(imgWts_ptr+ichan+irow*nDataChan)*abs(norm);
				    }
				}
			    } // End poln-loop
			}
		    }
		}
	    } // End chan-loop
	}
    } // End row-loop
  //exit(0);
}
//
//---------------------------------------------------------------------------------
//
void csgrid(Double pos[2], Int loc[3], Double off[3], Complex& phasor, 
	   const Int& irow, const Matrix<Double>& uvw, const Double& dphase, 
	   const Double& freq, const Double* scale, const Double* offset,
	   const Float sampling[2])
{
  Double phase;
  //Vector<Double> uvw_l(3,0); // This allows gridding of weights
  Double uvw_l[3]={0.0,0.0,0.0}; // This allows gridding of weights
  Bool dd;
  const Double *uvw_ptr=uvw.getStorage(dd);
  // centered on the uv-origin
  //  if (uvw.nelements() > 0) for(Int i=0;i<3;i++) uvw_l[i]=uvw(i,irow);
  if (uvw.nelements() > 0) for(Int i=0;i<3;i++) uvw_l[i]=uvw_ptr[i+irow*3];
  
  pos[2]=sqrt(abs(scale[2]*uvw_l[2]*freq/C::c))+offset[2];
  loc[2]=SynthesisUtils::nint(pos[2]);
  off[2]=0;
  
  for(Int idim=0;idim<2;idim++)
    {
      pos[idim]=scale[idim]*uvw_l[idim]*freq/C::c+(offset[idim]);
      loc[idim]=SynthesisUtils::nint(pos[idim]);
      //	off[idim]=SynthesisUtils::nint((loc[idim]-pos[idim])*sampling[idim]+1);
      off[idim]=SynthesisUtils::nint((loc[idim]-pos[idim])*sampling[idim]);
    }
  
  if (dphase != 0.0)
    {
      phase=-2.0*C::pi*dphase*freq/C::c;
      Double sp,cp;
      sincos(phase,&sp,&cp);
      //      phasor=Complex(cos(phase), sin(phase));
      phasor=Complex(cp,sp);
    }
  else
    phasor=Complex(1.0);
  // cerr << "### " << pos[0] << " " << offset[0] << " " << loc[0] << " " << off[0] << " " << uvw_l[0] << endl;
  // exit(0);
}
//
//---------------------------------------------------------------------------------
//
Bool ccomputeSupport(const VBStore* vbs, const Int& XThGrid, const Int& YThGrid,
		    const Int support[2], const Float sampling[2],
		    const Double pos[2], const Int loc[3],
		    Float iblc[2], Float itrc[2])
{
  //    Int sup[2] = {support[0]*sampling[0], support[1]*sampling[1]};
  Int sup[2] = {support[0], support[1]};
  Int blc[2] = {vbs->BLCXi(XThGrid, YThGrid), vbs->BLCYi(XThGrid, YThGrid)};
  Int trc[2] = {vbs->TRCXi(XThGrid, YThGrid), vbs->TRCYi(XThGrid, YThGrid)};

  Float vblc[2]={pos[0]-sup[0],pos[1]-sup[1]}, vtrc[2]={pos[0]+sup[0],pos[1]+sup[1]};
  if (SynthesisUtils::checkIntersection(blc,trc,vblc,vtrc))
    {
      SynthesisUtils::calcIntersection(blc,trc,vblc,vtrc,iblc,itrc);
      return True;
    }
  return False;
}
//
//---------------------------------------------------------------------------------
//
Complex* cgetConvFunc_p(Int cfShape[4], VBStore* vbs,
			Double& wVal, Int& fndx, Int& wndx,
			Int **mNdx, Int  **conjMNdx,
			Int& ipol, uInt& mRow)
{
  Bool Dummy;
  Complex *tt;
  CFCStruct *tcfc;
  Int polNdx, shape[3];

  if (wVal > 0.0) polNdx=mNdx[ipol][mRow];
  else            polNdx=conjMNdx[ipol][mRow];
  
  tcfc=vbs->cfBSt_p.getCFB(fndx,wndx,polNdx);
  
  tt=tcfc->CFCStorage;
  cfShape[0]=tcfc->shape[0];
  cfShape[1]=tcfc->shape[1];
  
  return tt;
};
//
//---------------------------------------------------------------------------------
//
void ccachePhaseGrad_g(Complex *cached_phaseGrad_p, Int phaseGradNX, Int phaseGradNY,
		       Double* cached_PointingOffset_p, Double* pointingOffset,
		       Int cfShape[4], Int convOrigin[4])
{
  if (
      ((fabs(pointingOffset[0]-cached_PointingOffset_p[0])) > 1e-6) ||
      ((fabs(pointingOffset[1]-cached_PointingOffset_p[1])) > 1e-6) ||
      (phaseGradNX < cfShape[0]) || (phaseGradNY < cfShape[1])
      )
    {
      cerr << "Computing phase gradiant for pointing offset " 
	   << "[" << pointingOffset[0] << "," << pointingOffset[1] << "] ["
	   << cfShape[0] << "," << cfShape[1] << "]" << endl;
      
      Int nx=cfShape[0], ny=cfShape[1];
      Double grad;
      Complex phx,phy;
      
      cerr << "Resize cached_phaseGrad_p !!!" << endl;
      //	cached_phaseGrad_p.resize(nx,ny);
      cached_PointingOffset_p[0] = pointingOffset[0];
      cached_PointingOffset_p[1] = pointingOffset[1];
      
      for(Int ix=0;ix<nx;ix++)
	{
	  grad = (ix-convOrigin[0])*pointingOffset[0];
	  Double sx,cx;
	  sincos(grad,&sx,&cx);
	  //	    phx = Complex(cos(grad),sin(grad));
	  phx = Complex(cx,sx);
	  for(Int iy=0;iy<ny;iy++)
	    {
	      grad = (iy-convOrigin[1])*pointingOffset[1];
	      Double sy,cy;
	      sincos(grad,&sy,&cy);
	      //		phy = Complex(cos(grad),sin(grad));
	      phy = Complex(cy,sy);
	      cached_phaseGrad_p[ix+iy*phaseGradNY]=phx*phy;
	    }
	}
    }
}
//
//---------------------------------------------------------------------------------
//
template <class T>
Complex caccumulateOnGrid(T* gridStore,
			  const Int* gridInc_p,
			  const Complex *cached_phaseGrad_p,
			  const Int cachedPhaseGradNX, const Int cachedPhaseGradNY,
			  const Complex* convFuncV, 
			  const Int *cfInc_p,
			  Complex nvalue,Double wVal, 
			  Int *supBLC_ptr, Int *supTRC_ptr,//Int* scaledSupport_ptr, 
			  Float* scaledSampling_ptr, 
			  Double* off_ptr, Int* convOrigin_ptr, 
			  Int* cfShape, Int* loc_ptr, Int* iGrdpos_ptr,
			  Bool finitePointingOffset,
			  Bool doPSFOnly, Bool& foundCFPeak)
{
  Int iloc_ptr[4]={0,0,0,0};//   for (int i=0;i<4;i++) iloc_ptr[i]=0;
  
  Complex wt, cfArea=1.0; 
  Complex norm=0.0;
  Int Nth = 1;
  
  Bool finitePointingOffset_l=finitePointingOffset;
  Bool doPSFOnly_l=doPSFOnly;
  Double wVal_l=wVal;
  Complex nvalue_l=nvalue;
  
  Int phaseGradOrigin_l[2]; 
  //    phaseGradOrigin_l = cached_phaseGrad_p.shape()/2;
  phaseGradOrigin_l[0] = cachedPhaseGradNX/2;
  phaseGradOrigin_l[1] = cachedPhaseGradNY/2;
  
  //    for(Int iy=-scaledSupport_ptr[1]; iy <= scaledSupport_ptr[1]; iy++) 
  for(Int iy=supBLC_ptr[1]; iy <= supTRC_ptr[1]; iy++) 
    {
      iloc_ptr[1]=(Int)((scaledSampling_ptr[1]*iy+off_ptr[1])-1)+convOrigin_ptr[1];
      iGrdpos_ptr[1]=loc_ptr[1]+iy;
      
      //	for(Int ix=-scaledSupport_ptr[0]; ix <= scaledSupport_ptr[0]; ix++) 
      for(Int ix=supBLC_ptr[0]; ix <= supTRC_ptr[0]; ix++) 
	{
	  iloc_ptr[0]=(Int)((scaledSampling_ptr[0]*ix+off_ptr[0])-1)+convOrigin_ptr[0];
	  iGrdpos_ptr[0]=loc_ptr[0]+ix;
	  {
	    if (ix==0 and iy==0) foundCFPeak=True;
	    wt = getFrom4DArray((const Complex * __restrict__ &)convFuncV, 
				iloc_ptr,cfInc_p)/cfArea;
	    if (wVal > 0.0) {wt = conj(wt);}
	    norm += (wt);
	    if (finitePointingOffset && !doPSFOnly) 
	      wt *= cached_phaseGrad_p[iloc_ptr[0]+phaseGradOrigin_l[0]+
				       iloc_ptr[1]+phaseGradOrigin_l[1]*cachedPhaseGradNY];
	    
	    // The following uses raw index on the 4D grid
	    addTo4DArray(gridStore,iGrdpos_ptr,gridInc_p, nvalue,wt);
	  }
	}
    }
  return norm;
}
#include "cDataToGridImpl_def.h"

};
