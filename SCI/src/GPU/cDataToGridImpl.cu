#include "hip/hip_runtime.h"
// -*- C++ -*-
//#include <synthesis/TransformMachines/cDataToGridImpl.h>
#include <casa/Arrays/Matrix.h>
//#include <hipfft/hipfft.h>
#include <hip/hip_complex.h>
#include "cDataToGridImpl.h"
#include <typeinfo>
#include <stdio.h>

namespace casa{

  __global__ void kernel_cuBlank(uInt *vbs,Int n)
  {
    printf("SubGrid: (%d %d) (%d %d) ", blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y);
  };

  void cuBlank(uInt* vbs,Int n)
  {
    Int NB=1, NT=1;
    dim3 dimBlock ( NB, NB, 1 ) ;
    dim3 dimThread( NT, NT, 1 ) ;
    
    //    kernel_cuBlank<<<dimBlock, dimThread>>>(vbs,n);
    hipDeviceSynchronize();
  }
  
  __global__ void kernel_cuDataToGridImpl_p(Complex* gridStore,  Int* gridShape, //4-elements
					    
					    const uInt *subGridShape,//[2],
					    const uInt *BLCXi, const uInt *BLCYi,
					    const uInt *TRCXi, const uInt *TRCYi,
					    
					    const Complex *visCube_ptr, const Float* imgWts_ptr,
					    const Bool *flagCube_ptr, const Bool *rowFlag_ptr,
					    const Double *uvw_ptr,
					    
					    const Int nRow, const Int rbeg, const Int rend, 
					    const Int nDataChan,const Int nDataPol, 
					    const Int startChan, const Int endChan, const Int vbSpw,
					    const Double *vbFreq,
					    
					    const Complex *cfV[2],
					    Int *cfShape,//[4], //[4]
					    Float *sampling,//[2], 
					    const Int *support, //[2]
					    
					    Double* sumWt_ptr,
					    const Bool dopsf, const Bool accumCFs,
					    const Int* polMap_ptr, const Int *chanMap_ptr,
					    const Double *uvwScale_ptr, const Double *offset_ptr,
					    const Double *dphase_ptr, Int XThGrid, Int YThGrid)
  {
    //printf("SubGrid: %d %d: %d %d %d %d %d %d %d %d\n", blockIdx.x, blockIdx.y,nRow,rbeg,rend,nDataChan,nDataPol,startChan,endChan,vbSpw);
    printf("SubGrid: %d %d: %d %d vis=%f %f\n", blockIdx.x, blockIdx.y, BLCXi[0], BLCXi[20],visCube_ptr[10],visCube_ptr[20]);
  };

  __global__ void kernel_cuDataToGridImpl_p(DComplex* gridStore,  Int* gridShape, //4-elements
					    
					    const uInt *subGridShape,//[2],
					    const uInt *BLCXi, const uInt *BLCYi,
					    const uInt *TRCXi, const uInt *TRCYi,
					    
					    const Complex *visCube_ptr, const Float* imgWts_ptr,
					    const Bool *flagCube_ptr, const Bool *rowFlag_ptr,
					    const Double *uvw_ptr,
					    
					    const Int nRow, const Int rbeg, const Int rend, 
					    const Int nDataChan,const Int nDataPol, 
					    const Int startChan, const Int endChan, const Int vbSpw,
					    const Double *vbFreq,
					    
					    const Complex *cfV[2],
					    Int *cfShape,//[4], //[4]
					    Float *sampling,//[2], 
					    const Int *support, //[2]
					    
					    Double* sumWt_ptr,
					    const Bool dopsf, const Bool accumCFs,
					    const Int* polMap_ptr, const Int *chanMap_ptr,
					    const Double *uvwScale_ptr, const Double *offset_ptr,
					    const Double *dphase_ptr, Int XThGrid, Int YThGrid)
  {
    //    printf("DSubGrid: %d %d: %d %d %d %d %d %d %d %d\n", blockIdx.x, blockIdx.y,nRow,rbeg,rend,nDataChan,nDataPol,startChan,endChan,vbSpw);
    printf("DSubGrid: %d %d: %d %d vis=%f %f %d %d %d %d %d %d %d\n", blockIdx.x, blockIdx.y, BLCXi[0], BLCXi[20], 
	   ((hipComplex*)visCube_ptr)[10].x,((hipComplex *)visCube_ptr)[20].x,nRow, nDataChan, nDataPol,
	   cfShape[0], cfShape[1], cfShape[2], cfShape[3]);
  };
  
  template <class T>
  void cuDataToGridImpl_p(T* gridStore,  Int* gridShape, //4-elements
			  
  			  const uInt *subGridShape,//[2],
  			  const uInt *BLCXi, const uInt *BLCYi,
  			  const uInt *TRCXi, const uInt *TRCYi,
			  
  			  const Complex *visCube_ptr, const Float* imgWts_ptr,
  			  const Bool *flagCube_ptr, const Bool *rowFlag_ptr,
  			  const Double *uvw_ptr,
			  
  			  const Int nRow, const Int rbeg, const Int rend, 
  			  const Int nDataChan,const Int nDataPol, 
  			  const Int startChan, const Int endChan, const Int vbSpw,
  			  const Double *vbFreq,
			  
			  const Complex *cfV[2],
  			  Int *cfShape,//[4], //[4]
  			  Float *sampling,//[2], 
			  const Int *support, //[2]
			  
  			  Double* sumWt_ptr,
  			  const Bool dopsf, const Bool accumCFs,
  			  const Int* polMap_ptr, const Int *chanMap_ptr,
  			  const Double *uvwScale_ptr, const Double *offset_ptr,
  			  const Double *dphase_ptr, Int XThGrid, Int YThGrid)
  {
    static Int tt=0;
    Int NB=1, NT=1;
    dim3 dimBlock ( NB, NB, 1 ) ;
    dim3 dimThread( NT, NT, 1 ) ;
    
    // kernel_cuDataToGridImpl_p<<<dimBlock,dimThread>>>(gridStore, gridShape, vbs, sumwt, dopsf, polMap_ptr, chanMap_ptr,
    // 						      uvwScale_ptr, offset_ptr, dphase_ptr, XThGrid, YThGrid);
    
    //    cerr << BLCXi << endl;

    // kernel_cuDataToGridImpl_p<<<dimBlock,dimThread>>>(gridStore, gridShape,subGridShape,BLCXi,BLCYi,TRCXi,TRCYi,
    // 						      visCube_ptr,imgWts_ptr,flagCube_ptr,rowFlag_ptr,uvw_ptr,
						      
    // 						      nRow,rbeg,rend,nDataChan,nDataPol,startChan,endChan,vbSpw,
    // 						      vbFreq,
						      
    // 						      cfV,cfShape,sampling,support,
		       
    // 						      sumWt_ptr,dopsf,accumCFs,polMap_ptr,chanMap_ptr,
    // 						      uvwScale_ptr,offset_ptr,dphase_ptr,XThGrid,YThGrid);
    // hipDeviceSynchronize();
    // if (tt++ > 10) exit(0);

    cDataToGridImpl2_p(gridStore, gridShape,subGridShape,BLCXi,BLCYi,TRCXi,TRCYi,
    		       visCube_ptr,imgWts_ptr,flagCube_ptr,rowFlag_ptr,uvw_ptr,
		       
    		       nRow,rbeg,rend,nDataChan,nDataPol,startChan,endChan,vbSpw,
    		       vbFreq,
		       
    		       cfV,cfShape,sampling,support,
		       
    		       sumWt_ptr,dopsf,accumCFs,polMap_ptr,chanMap_ptr,
    		       uvwScale_ptr,offset_ptr,dphase_ptr,XThGrid,YThGrid);
    
    hipError_t err=hipGetLastError();
    if (err != hipSuccess)
      {
	cerr << "###Cuda error: Failed to run the kernel " << hipGetErrorString (err) << endl;
	exit(0);
      }
  };
  
  template <class T>
  void cDataToGridImpl2_p(T* gridStore,  Int* gridShape, //4-elements
			  
  			  const uInt *subGridShape,//[2],
  			  const uInt *BLCXi, const uInt *BLCYi,
  			  const uInt *TRCXi, const uInt *TRCYi,
			  
  			  const Complex *visCube_ptr, const Float* imgWts_ptr,
  			  const Bool *flagCube_ptr, const Bool *rowFlag_ptr,
  			  const Double *uvw_ptr,
			  
  			  const Int nRow, const Int rbeg, const Int rend, 
  			  const Int nDataChan,const Int nDataPol, 
  			  const Int startChan, const Int endChan, const Int vbSpw,
  			  const Double *vbFreq,
			  
			  const Complex *cfV[2],
  			  Int *cfShape,//[4], //[4]
  			  Float *sampling,//[2], 
			  const Int *support, //[2]
			  
  			  Double* sumWt_ptr,
  			  const Bool dopsf, const Bool accumCFs,
  			  const Int* polMap_ptr, const Int *chanMap_ptr,
  			  const Double *uvwScale_ptr, const Double *offset_ptr,
  			  const Double *dphase_ptr, Int XThGrid, Int YThGrid)
  {
    //LogIO log_l(LogOrigin("ProtoVR[R&D]","DataToGridImpl_p"));
    
    //    Complex *cfV[2];
    
    Int nw, nCFFreq, nx,ny, nGridPol, nGridChan;
    Int targetIMChan, targetIMPol;
    
    Int loc[3], iloc[4],tiloc[4];
    Int convOrigin[4], gridInc_l[4], cfInc_l[4]; 
    Double pos[2], off[3];
    Int igrdpos[4];
    
    Complex phasor, nvalue, wt;
    Complex norm;
    Bool Dummy;
    // Bool * flagCube_ptr=vbs->flagCube_p.getStorage(Dummy);
    // Bool * rowFlag_ptr = vbs->rowFlag_p.getStorage(Dummy);
    // Float * imgWts_ptr = vbs->imagingWeight_p.getStorage(Dummy);
    // Complex * visCube_ptr = vbs->visCube_p.getStorage(Dummy);
    
    Double *pointingOffset_ptr=NULL; //vbs->cfBSt_p.pointingOffset
    Double *cached_PointingOffset_ptr=NULL;
    
    
    nx=gridShape[0]; ny=gridShape[1];
    nGridPol=gridShape[2]; nGridChan=gridShape[3];
    Bool gDummy;
    
    //    Double *freq=vbs->freq_p.getStorage(Dummy);
    
    cacheAxisIncrements(gridShape, gridInc_l);
    
    // nCFFreq = vbs->cfBSt_p.shape[0]; // shape[0]: nChan, shape[1]: nW, shape[2]: nPol
    // nw = vbs->cfBSt_p.shape[1];
    
    iloc[0]=iloc[1]=iloc[2]=iloc[3]=0;
    
    // if (accumCFs)
    //   {
    // 	startChan = vbs->startChan_p;
    // 	endChan = vbs->endChan_p;
    //   }
    // else 
    //   {
    // 	startChan = 0;
    // 	endChan = vbs->nDataChan_p;
    //   }
    
    Bool finitePointingOffsets = False;
    for(Int irow=rbeg; irow< rend; irow++)
      {   
  	if(!(*(rowFlag_ptr+irow)))
  	  {   
  	    for(Int ichan=startChan; ichan< endChan; ichan++)
  	      {
  		if (*(imgWts_ptr + ichan+irow*nDataChan)!=0.0) 
  		  {  
  		    targetIMChan=chanMap_ptr[ichan];
		    
  		    if((targetIMChan>=0) && (targetIMChan<nGridChan)) 
  		      {
  			// Double dataWVal = vbs->vb_p->uvw()(irow)(2);
  			Double dataWVal = 0;
  			//			if (vbs->uvw_p.nelements() > 0) dataWVal = vbs->uvw_p(irow,2);
  			if (uvw_ptr != NULL) dataWVal = uvw_ptr[irow+nRow*2];
			
  			Int wndx = 0;//(int)(sqrt(vbs->cfBSt_p.wIncr*abs(dataWVal*vbFreq[ichan]/C::c)));
			
  			Int cfFreqNdx=0;
  			// if (vbs->conjBeams_p) cfFreqNdx = vbs->cfBSt_p.conjFreqNdxMap[vbSpw][ichan];
  			// else cfFreqNdx = vbs->cfBSt_p.freqNdxMap[vbSpw][ichan];
			
  			Float s;
  			// s=vbs->cfBSt_p.CFBStorage->sampling;
  			// support[0]=vbs->cfBSt_p.CFBStorage->xSupport;
  			// support[1]=vbs->cfBSt_p.CFBStorage->ySupport;
			
  			// sampling[0] = sampling[1] = SynthesisUtils::nint(s);
			
  			//			const Double *uvw_ptr=vbs->uvw_p.getStorage(Dummy);
			
  			// *uvwScale_ptr=uvwScale_p.getStorage(Dummy),
  			// *offset_ptr=offset_p.getStorage(Dummy);;
			
  			csgrid(pos,loc,off, phasor, irow, uvw_ptr, dphase_ptr[irow], vbFreq[ichan], 
  			       uvwScale_ptr, offset_ptr, sampling);
			
  			Float cfblc[2], cftrc[2];
  			//		    pos[0]=1024.1;pos[1]=1025.6;
  			Bool onMyGrid=
			  ccomputeSupport(BLCXi,BLCYi, TRCXi, TRCYi, subGridShape,
					  XThGrid, YThGrid, support, sampling, pos, loc,cfblc,cftrc);
			//			onMyGrid=ccomputeSupport(vbs, XThGrid, YThGrid, support, sampling, pos, loc,cfblc,cftrc);
  			// 		    if (onGrid(nx, ny, nw, loc, support)) 
  			if (onMyGrid)
  			  {
  			    Int iblc[2], itrc[2];
			    
  			    iblc[0]=SynthesisUtils::nint((cfblc[0]-pos[0]));///sampling[0]);
  			    iblc[1]=SynthesisUtils::nint((cfblc[1]-pos[1]));///sampling[1]);
  			    itrc[0]=SynthesisUtils::nint((cftrc[0]-pos[0]));///sampling[0]);
  			    itrc[1]=SynthesisUtils::nint((cftrc[1]-pos[1]));///sampling[1]); 
			    
  			    Int dx=abs(itrc[0]-iblc[0])+1, dy=abs(itrc[1]-iblc[1])+1;
  			    Float cfFractioanlArea = (dx*dy)/(float)square(abs(support[1]+support[0])+1);
			    
  			    // Loop over all image-plane polarization planes.
  			    for(Int ipol=0; ipol< nDataPol; ipol++) 
  			      { 
  				if((!(*(flagCube_ptr + ipol + ichan*nDataPol + irow*nDataPol*nDataChan))))
  				  {  
  				    targetIMPol=polMap_ptr[ipol];
  				    if ((targetIMPol>=0) && (targetIMPol<nGridPol)) 
  				      {
  					igrdpos[2]=targetIMPol; igrdpos[3]=targetIMChan;
					
  					// if(accumCFs)     allPolNChanDone_l(ipol,ichan,irow)=True;
  					if(dopsf) nvalue=Complex(*(imgWts_ptr + ichan + irow*nDataChan));
  					else      nvalue= *(imgWts_ptr+ichan+irow*nDataChan)*
  						    (*(visCube_ptr+ipol+ichan*nDataPol+irow*nDataChan*nDataPol)*phasor);
					
  					norm = 0.0;
  					Bool foundCFPeak=False;
  					uInt nMueller=1; //vbs->cfBSt_p.nMueller
  					for (uInt mRow=0;mRow<nMueller; mRow++) 
  					  {
  					    const Complex* convFuncV;
  					    Int muellerElementsIndex[4][1] ={{0},{},{},{1}};
  					    Int conjMuellerElementsIndex[4][1] ={{1},{},{},{0}};
					    Int polNdx;
					    if (dataWVal > 0.0) polNdx=muellerElementsIndex[ipol][mRow];
					    else                polNdx=conjMuellerElementsIndex[ipol][mRow];
					    convFuncV = cfV[polNdx];
					    
  					    // convFuncV=cgetConvFunc_p(cfShape, vbs, dataWVal, cfFreqNdx, wndx, 
  					    // 			     // vbs->cfBSt_p.muellerElementsIndex,
  					    // 			     // vbs->cfBSt_p.conjMuellerElementsIndex, 
  					    // 			     muellerElementsIndex, conjMuellerElementsIndex,
  					    // 			     ipol,  mRow);
					    
  					    convOrigin[0]=cfShape[0]/2;
  					    convOrigin[1]=cfShape[1]/2;
  					    convOrigin[2]=cfShape[2]/2;
  					    convOrigin[3]=cfShape[3]/2;
  					    Bool psfOnly=((dopsf==True) && (accumCFs==False));
					    
  					    Int cachedPhaseGradNX=0,cachedPhaseGradNY=0;
  					    Complex *cached_PhaseGrad_ptr=NULL;
					    
  					    if (finitePointingOffsets && !psfOnly)
  					      ccachePhaseGrad_g(cached_PhaseGrad_ptr, cachedPhaseGradNX, cachedPhaseGradNY,	
  								cached_PointingOffset_ptr, pointingOffset_ptr, cfShape, convOrigin);//, cfRefFreq);//, vbs->imRefFreq());
					    
  					    cacheAxisIncrements(cfShape, cfInc_l);
  					    norm += caccumulateOnGrid(gridStore, gridInc_l, cached_PhaseGrad_ptr, 
  								      cachedPhaseGradNX, cachedPhaseGradNY,
  								      convFuncV, cfInc_l, nvalue,dataWVal,
  								      iblc, itrc, sampling, off, 
  								      convOrigin, cfShape, loc, igrdpos,
  								      finitePointingOffsets, psfOnly, foundCFPeak);
  					  }
					
  					*(sumWt_ptr+targetIMPol+targetIMChan*nGridPol)+= *(imgWts_ptr+ichan+irow*nDataChan)*abs(norm);
  				      }
  				  }
  			      } // End poln-loop
  			  }
  		      }
  		  }
  	      } // End chan-loop
  	  }
      } // End row-loop
    //exit(0);
  }
  
  template <class T>
  void cDataToGridImpl_p(T* gridStore,  Int* gridShape /*4-elements*/,
			 VBStore* vbs, Matrix<Double>* sumwt,
			 const Bool dopsf,
			 const Int* polMap_ptr, const Int *chanMap_ptr,
			 const Double *uvwScale_ptr, const Double *offset_ptr,
			 const Double *dphase_ptr, Int XThGrid, Int YThGrid)
  {
    //LogIO log_l(LogOrigin("ProtoVR[R&D]","DataToGridImpl_p"));
    
    // Complex tmp;
    // Bool isGridSinglePrecision=(typeid(gridStore[0]) == typeid(tmp));
    // cerr << "cuisGridSinglePrecision = " << isGridSinglePrecision << endl;
    
    
    Int nGridPol, nGridChan, nx, ny, nw, nCFFreq;
    Int targetIMChan, targetIMPol, rbeg, rend;
    Int startChan, endChan;
    Bool accumCFs;
    
    Float sampling[2],scaledSampling[2];
    Int support[2],loc[3], iloc[4],tiloc[4],scaledSupport[2];
    Int convOrigin[4], gridInc_l[4], cfInc_l[4]; 
    Double pos[2], off[3];
    Int igrdpos[4];
    
    Complex phasor, nvalue, wt;
    Complex norm;
    Int cfShape[4];
    Bool Dummy;
    Bool * flagCube_ptr=vbs->flagCube_p.getStorage(Dummy);
    Bool * rowFlag_ptr = vbs->rowFlag_p.getStorage(Dummy);
    Float * imgWts_ptr = vbs->imagingWeight_p.getStorage(Dummy);
    Complex * visCube_ptr = vbs->visCube_p.getStorage(Dummy);
    Double *sumWt_ptr=sumwt->getStorage(Dummy);
    
    //  Vector<Double> pointingOffset(cfb.getPointingOffset());
    // Double *pointingOffset_ptr=vbs->cfBSt_p.pointingOffset,
    //   *cached_PointingOffset_ptr=cached_PointingOffset_p.getStorage(Dummy);
    Double *pointingOffset_ptr=vbs->cfBSt_p.pointingOffset,
      *cached_PointingOffset_ptr=NULL;
    
    //    cerr << "Data_dptr = " << vbs->visCube_dptr << endl;
    
    
    Int vbSpw=vbs->spwID_p;
    
    
    rbeg = vbs->beginRow_p;
    rend = vbs->endRow_p;
    
    nx=gridShape[0]; ny=gridShape[1];
    nGridPol=gridShape[2]; nGridChan=gridShape[3];
    Bool gDummy;
    
    Double *freq=vbs->freq_p.getStorage(Dummy);
    
    cacheAxisIncrements(gridShape, gridInc_l);
    
    nCFFreq = vbs->cfBSt_p.shape[0]; // shape[0]: nChan, shape[1]: nW, shape[2]: nPol
    nw = vbs->cfBSt_p.shape[1];
    
    iloc[0]=iloc[1]=iloc[2]=iloc[3]=0;
    Int nDataChan=vbs->nDataChan_p,
      nDataPol = vbs->nDataPol_p;
    accumCFs=vbs->accumCFs_p;
    if (accumCFs)
      {
	startChan = vbs->startChan_p;
	endChan = vbs->endChan_p;
      }
    else 
      {
	startChan = 0;
	endChan = vbs->nDataChan_p;
      }
    
    
    //  cerr << "ProtoVR: " << rbeg << " " << rend << " " << startChan << " " << endChan << " " << nDataChan << " " << nDataPol << endl;
    
    // Bool finitePointingOffsets= (
    // 			       (fabs(pointingOffset_ptr[0])>0) ||  
    // 			       (fabs(pointingOffset_ptr[1])>0)
    // 			       );
    Bool finitePointingOffsets = False;
    for(Int irow=rbeg; irow< rend; irow++)
      {   
	if(!(*(rowFlag_ptr+irow)))
	  {   
	    for(Int ichan=startChan; ichan< endChan; ichan++)
	      {
		if (*(imgWts_ptr + ichan+irow*nDataChan)!=0.0) 
		  {  
		    targetIMChan=chanMap_ptr[ichan];
		    
		    if((targetIMChan>=0) && (targetIMChan<nGridChan)) 
		      {
			// Double dataWVal = vbs->vb_p->uvw()(irow)(2);
			Double dataWVal = 0;
			if (vbs->uvw_p.nelements() > 0) dataWVal = vbs->uvw_p(irow,2);
			
			Int wndx = (int)(sqrt(vbs->cfBSt_p.wIncr*abs(dataWVal*freq[ichan]/C::c)));
			
			Int cfFreqNdx;
			if (vbs->conjBeams_p) cfFreqNdx = vbs->cfBSt_p.conjFreqNdxMap[vbSpw][ichan];
			else cfFreqNdx = vbs->cfBSt_p.freqNdxMap[vbSpw][ichan];
			
			Float s;
			s=vbs->cfBSt_p.CFBStorage->sampling;
			support[0]=vbs->cfBSt_p.CFBStorage->xSupport;
			support[1]=vbs->cfBSt_p.CFBStorage->ySupport;
			
			sampling[0] = sampling[1] = SynthesisUtils::nint(s);
			
			const Double *uvw_ptr=NULL;
			if (vbs->uvw_p.nelements() > 0) uvw_ptr=vbs->uvw_p.getStorage(Dummy);
			// *uvwScale_ptr=uvwScale_p.getStorage(Dummy),
			// *offset_ptr=offset_p.getStorage(Dummy);;
			
			csgrid(pos,loc,off, phasor, irow, uvw_ptr, dphase_ptr[irow], freq[ichan], 
			       uvwScale_ptr, offset_ptr, sampling);
			
			Float cfblc[2], cftrc[2];
			//		    pos[0]=1024.1;pos[1]=1025.6;
			
			uInt subGridShape[2]={vbs->BLCXi.shape()(0), vbs->BLCXi.shape()(1)};
			const uInt *BLCXi_ptr=vbs->BLCXi.getStorage(Dummy);
			const uInt *BLCYi_ptr=vbs->BLCYi.getStorage(Dummy);
			const uInt *TRCXi_ptr=vbs->TRCXi.getStorage(Dummy);
			const uInt *TRCYi_ptr=vbs->TRCYi.getStorage(Dummy);
			Bool onMyGrid=
			  ccomputeSupport(BLCXi_ptr,BLCYi_ptr, TRCXi_ptr, TRCYi_ptr, subGridShape,
					  XThGrid, YThGrid, support, sampling, pos, loc,cfblc,cftrc);
			// ccomputeSupport(vbs, XThGrid, YThGrid, support, sampling, pos, loc,cfblc,cftrc);
			// 		    if (onGrid(nx, ny, nw, loc, support)) 
			if (onMyGrid)
			  {
			    
			    Int iblc[2], itrc[2];
			    
			    iblc[0]=SynthesisUtils::nint((cfblc[0]-pos[0]));///sampling[0]);
			    iblc[1]=SynthesisUtils::nint((cfblc[1]-pos[1]));///sampling[1]);
			    itrc[0]=SynthesisUtils::nint((cftrc[0]-pos[0]));///sampling[0]);
			    itrc[1]=SynthesisUtils::nint((cftrc[1]-pos[1]));///sampling[1]); 
			    
			    Int dx=abs(itrc[0]-iblc[0])+1, dy=abs(itrc[1]-iblc[1])+1;
			    Float cfFractioanlArea = (dx*dy)/(float)square(abs(support[1]+support[0])+1);
			    
			    // if ((irow < 3) && (ichan == 2))
			    //   {
			    //     // cerr << irow << "#[" << XThGrid << "," << YThGrid << "] " 
			    //     // 	 << (cfblc[0]) << " " << (cftrc[0]) << " " 
			    //     // 	 << (cfblc[1]) << " " << (cftrc[1]) << " " 
			    //     // 	 << dx << " " << dy << " " << (dx*dy) << " " << cfFractionalArea
			    //     // 	 << endl;
			    //     // cerr << irow << "#[" << XThGrid << "," << YThGrid << "] " 
			    //     // 	 << (pos[0]) << " " << (pos[1]) << endl;
			    
			    //     cerr << irow << " [" << XThGrid << "," << YThGrid << "] " 
			    // 	   << iblc[0] << " " << itrc[0] << " " 
			    // 	   << iblc[1] << " " << itrc[1] << " " << rend
			    // 	   << endl;
			    //   }
			    
			    // Loop over all image-plane polarization planes.
			    for(Int ipol=0; ipol< nDataPol; ipol++) 
			      { 
				if((!(*(flagCube_ptr + ipol + ichan*nDataPol + irow*nDataPol*nDataChan))))
				  {  
				    targetIMPol=polMap_ptr[ipol];
				    if ((targetIMPol>=0) && (targetIMPol<nGridPol)) 
				      {
					igrdpos[2]=targetIMPol; igrdpos[3]=targetIMChan;
					
					// if(accumCFs)     allPolNChanDone_l(ipol,ichan,irow)=True;
					if(dopsf) nvalue=Complex(*(imgWts_ptr + ichan + irow*nDataChan));
					else      nvalue= *(imgWts_ptr+ichan+irow*nDataChan)*
						    (*(visCube_ptr+ipol+ichan*nDataPol+irow*nDataChan*nDataPol)*phasor);
					
					norm = 0.0;
					// for (uInt mRow=0;mRow<conjMNdx[ipol].nelements(); mRow++) 
					// for (uInt mRow=0;mRow<vbs.cfBSt_p.conjMuellerElementsIndex[ipol].nelements(); mRow++) 
					Bool foundCFPeak=False;
					for (uInt mRow=0;mRow<vbs->cfBSt_p.nMueller; mRow++) 
					  {
					    Complex* convFuncV;
					    // CUWORK:  Essentially CFC.getCellPtr(FNDX, WNDX, POLNDX)
					    // CUWORK: CFC wrapper
					    Int muellerElementsIndex[4][1]     = {{0},{},{},{1}};
					    Int conjMuellerElementsIndex[4][1] = {{1},{},{},{0}};
					    
					    convFuncV=cgetConvFunc_p(cfShape, vbs, dataWVal, cfFreqNdx, wndx, 
								     // vbs->cfBSt_p.muellerElementsIndex,
								     // vbs->cfBSt_p.conjMuellerElementsIndex, 
								     muellerElementsIndex, conjMuellerElementsIndex,
								     ipol,  mRow);
					    
					    convOrigin[0]=cfShape[0]/2;
					    convOrigin[1]=cfShape[1]/2;
					    convOrigin[2]=cfShape[2]/2;
					    convOrigin[3]=cfShape[3]/2;
					    Bool psfOnly=((dopsf==True) && (accumCFs==False));
					    // // CUWORK: Convert to a global function with native types
					    
					    // Int cachedPhaseGradNX=cached_phaseGrad_p.shape()[0],
					    //   cachedPhaseGradNY=cached_phaseGrad_p.shape()[1];
					    // Complex *cached_PhaseGrad_ptr=cached_phaseGrad_p.getStorage(Dummy);
					    Int cachedPhaseGradNX=0,cachedPhaseGradNY=0;
					    Complex *cached_PhaseGrad_ptr=NULL;
					    
					    if (finitePointingOffsets && !psfOnly)
					      ccachePhaseGrad_g(cached_PhaseGrad_ptr, cachedPhaseGradNX, cachedPhaseGradNY,	
								cached_PointingOffset_ptr, pointingOffset_ptr, cfShape, convOrigin);//, cfRefFreq);//, vbs->imRefFreq());
					    
					    cacheAxisIncrements(cfShape, cfInc_l);
					    //cerr << gridShape[0] << " " << gridShape[1] << " " << gridInc_p[0] << " " << gridInc_p[0] << endl;
					    norm += caccumulateOnGrid(gridStore, gridInc_l, cached_PhaseGrad_ptr, 
								      cachedPhaseGradNX, cachedPhaseGradNY,
								      convFuncV, cfInc_l, nvalue,dataWVal,
								      iblc, itrc,/*support,*/ sampling, off, 
								      convOrigin, cfShape, loc, igrdpos,
								      finitePointingOffsets,psfOnly,foundCFPeak);
					  }
					
					//sumwt(targetIMPol,targetIMChan) += vbs->imagingWeight_p(ichan, irow);//*abs(norm);
					//cerr << sumwt << " " << targetIMPol << " " << targetIMChan << " " << vbs->imagingWeight_p(ichan, irow) << " " << abs(norm) << endl;
					// Int dx=abs(itrc[0]-iblc[0]+1), dy=abs(itrc[1]-iblc[1]+1);
					// Float cfPixArea = (float)square(abs(support[0]-support[1]+1));
					
					// Accumulate data weight only when the CF peak was used. This can also be done via
					// fractional area of the CF used, and probably should be done that for high accuracy 
					// using the norm of the CF.
					//if (foundCFPeak) 
					*(sumWt_ptr+targetIMPol+targetIMChan*nGridPol)+= *(imgWts_ptr+ichan+irow*nDataChan)*abs(norm);
				      }
				  }
			      } // End poln-loop
			  }
		      }
		  }
	      } // End chan-loop
	  }
      } // End row-loop
    //exit(0);
  }
  //
  //---------------------------------------------------------------------------------
  //
  void csgrid(Double pos[2], Int loc[3], Double off[3], Complex& phasor, 
	      const Int& irow, const Double* uvw_ptr, const Double& dphase, 
	      const Double& freq, const Double* scale, const Double* offset,
	      const Float sampling[2])
  {
    Double phase;
    //Vector<Double> uvw_l(3,0); // This allows gridding of weights
    Double uvw_l[3]={0.0,0.0,0.0}; // This allows gridding of weights
    
    //    const Double *uvw_ptr=uvw.getStorage(dd);
    // centered on the uv-origin
    //  if (uvw.nelements() > 0) for(Int i=0;i<3;i++) uvw_l[i]=uvw(i,irow);
    // if (uvw.nelements() > 0) for(Int i=0;i<3;i++) uvw_l[i]=uvw_ptr[i+irow*3];
    if (uvw_ptr != NULL) for(Int i=0;i<3;i++) uvw_l[i]=uvw_ptr[i+irow*3];
    
    pos[2]=sqrt(abs(scale[2]*uvw_l[2]*freq/C::c))+offset[2];
    loc[2]=SynthesisUtils::nint(pos[2]);
    off[2]=0;
    
    for(Int idim=0;idim<2;idim++)
      {
	pos[idim]=scale[idim]*uvw_l[idim]*freq/C::c+(offset[idim]);
	loc[idim]=SynthesisUtils::nint(pos[idim]);
	//	off[idim]=SynthesisUtils::nint((loc[idim]-pos[idim])*sampling[idim]+1);
	off[idim]=SynthesisUtils::nint((loc[idim]-pos[idim])*sampling[idim]);
      }
    
    if (dphase != 0.0)
      {
	phase=-2.0*C::pi*dphase*freq/C::c;
	Double sp,cp;
	sincos(phase,&sp,&cp);
	//      phasor=Complex(cos(phase), sin(phase));
	phasor=Complex(cp,sp);
      }
    else
      phasor=Complex(1.0);
    // cerr << "### " << pos[0] << " " << offset[0] << " " << loc[0] << " " << off[0] << " " << uvw_l[0] << endl;
    // exit(0);
  }
  //
  //---------------------------------------------------------------------------------
  //
  Bool ccomputeSupport(const uInt *BLCXi_ptr, const uInt *BLCYi_ptr,
		       const uInt *TRCXi_ptr, const uInt *TRCYi_ptr,
		       const uInt subGridShape[2],
		       const Int& XThGrid, const Int& YThGrid,
		       const Int support[2], const Float sampling[2],
		       const Double pos[2], const Int loc[3],
		       Float iblc[2], Float itrc[2])
  {
    //    Int sup[2] = {support[0]*sampling[0], support[1]*sampling[1]};
    Int sup[2] = {support[0], support[1]};
    // Int subGridShape[2]={vbs->BLCXi.shape()(0), vbs->BLCXi.shape()(1)};
    // Bool Dummy;
    // const uInt *BLCXi_ptr=vbs->BLCXi.getStorage(Dummy);
    // const uInt *BLCYi_ptr=vbs->BLCYi.getStorage(Dummy);
    // const uInt *TRCXi_ptr=vbs->TRCXi.getStorage(Dummy);
    // const uInt *TRCYi_ptr=vbs->TRCYi.getStorage(Dummy);
    
    Int blc[2] = {BLCXi_ptr[XThGrid + YThGrid*subGridShape[0]], BLCYi_ptr[XThGrid + YThGrid*subGridShape[0]]};
    Int trc[2] = {TRCXi_ptr[XThGrid + YThGrid*subGridShape[0]], TRCYi_ptr[XThGrid + YThGrid*subGridShape[0]]};
    
    // Int blc[2] = {vbs->BLCXi(XThGrid, YThGrid), vbs->BLCYi(XThGrid, YThGrid)};
    // Int trc[2] = {vbs->TRCXi(XThGrid, YThGrid), vbs->TRCYi(XThGrid, YThGrid)};
    
    Float vblc[2]={pos[0]-sup[0],pos[1]-sup[1]}, vtrc[2]={pos[0]+sup[0],pos[1]+sup[1]};

    if (SynthesisUtils::checkIntersection(blc,trc,vblc,vtrc))
      {
	SynthesisUtils::calcIntersection(blc,trc,vblc,vtrc,iblc,itrc);
	return True;
      }
    return False;
  }
  //
  //---------------------------------------------------------------------------------
  //
  Complex* cgetConvFunc_p(Int cfShape[4], VBStore* vbs,
			  Double& wVal, Int& fndx, Int& wndx,
			  //Int **mNdx, Int  **conjMNdx,
			  Int mNdx[4][1], Int conjMNdx[4][1],
			  Int& ipol, uInt& mRow)
  {
    Bool Dummy;
    Complex *tt;
    CFCStruct *tcfc;
    Int polNdx, shape[3];
    
    if (wVal > 0.0) polNdx=mNdx[ipol][mRow];
    else            polNdx=conjMNdx[ipol][mRow];
    
    tcfc=vbs->cfBSt_p.getCFB(fndx,wndx,polNdx);
    
    tt=tcfc->CFCStorage;
    cfShape[0]=tcfc->shape[0];
    cfShape[1]=tcfc->shape[1];
    
    return tt;
  };
  //
  //---------------------------------------------------------------------------------
  //
  void ccachePhaseGrad_g(Complex *cached_phaseGrad_p, Int phaseGradNX, Int phaseGradNY,
			 Double* cached_PointingOffset_p, Double* pointingOffset,
			 Int cfShape[4], Int convOrigin[4])
  {
    if (
	((fabs(pointingOffset[0]-cached_PointingOffset_p[0])) > 1e-6) ||
	((fabs(pointingOffset[1]-cached_PointingOffset_p[1])) > 1e-6) ||
	(phaseGradNX < cfShape[0]) || (phaseGradNY < cfShape[1])
	)
      {
	cerr << "Computing phase gradiant for pointing offset " 
	     << "[" << pointingOffset[0] << "," << pointingOffset[1] << "] ["
	     << cfShape[0] << "," << cfShape[1] << "]" << endl;
	
	Int nx=cfShape[0], ny=cfShape[1];
	Double grad;
	Complex phx,phy;
	
	cerr << "Resize cached_phaseGrad_p !!!" << endl;
	//	cached_phaseGrad_p.resize(nx,ny);
	cached_PointingOffset_p[0] = pointingOffset[0];
	cached_PointingOffset_p[1] = pointingOffset[1];
	
	for(Int ix=0;ix<nx;ix++)
	  {
	    grad = (ix-convOrigin[0])*pointingOffset[0];
	    Double sx,cx;
	    sincos(grad,&sx,&cx);
	    //	    phx = Complex(cos(grad),sin(grad));
	    phx = Complex(cx,sx);
	    for(Int iy=0;iy<ny;iy++)
	      {
		grad = (iy-convOrigin[1])*pointingOffset[1];
		Double sy,cy;
		sincos(grad,&sy,&cy);
		//		phy = Complex(cos(grad),sin(grad));
		phy = Complex(cy,sy);
		cached_phaseGrad_p[ix+iy*phaseGradNY]=phx*phy;
	      }
	  }
      }
  }
  //
  //---------------------------------------------------------------------------------
  //
  template <class T>
  Complex caccumulateOnGrid(T* gridStore,
			    const Int* gridInc_p,
			    const Complex *cached_phaseGrad_p,
			    const Int cachedPhaseGradNX, const Int cachedPhaseGradNY,
			    const Complex* convFuncV, 
			    const Int *cfInc_p,
			    Complex nvalue,Double wVal, 
			    Int *supBLC_ptr, Int *supTRC_ptr,//Int* scaledSupport_ptr, 
			    Float* scaledSampling_ptr, 
			    Double* off_ptr, Int* convOrigin_ptr, 
			    Int* cfShape, Int* loc_ptr, Int* iGrdpos_ptr,
			    Bool finitePointingOffset,
			    Bool doPSFOnly, Bool& foundCFPeak)
  {
    Int iloc_ptr[4]={0,0,0,0};//   for (int i=0;i<4;i++) iloc_ptr[i]=0;
    
    Complex wt, cfArea=1.0; 
    Complex norm=0.0;
    Int Nth = 1;
    
    Bool finitePointingOffset_l=finitePointingOffset;
    Bool doPSFOnly_l=doPSFOnly;
    Double wVal_l=wVal;
    Complex nvalue_l=nvalue;
    
    Int phaseGradOrigin_l[2]; 
    //    phaseGradOrigin_l = cached_phaseGrad_p.shape()/2;
    phaseGradOrigin_l[0] = cachedPhaseGradNX/2;
    phaseGradOrigin_l[1] = cachedPhaseGradNY/2;
    
    //    for(Int iy=-scaledSupport_ptr[1]; iy <= scaledSupport_ptr[1]; iy++) 
    for(Int iy=supBLC_ptr[1]; iy <= supTRC_ptr[1]; iy++) 
      {
	iloc_ptr[1]=(Int)((scaledSampling_ptr[1]*iy+off_ptr[1])-1)+convOrigin_ptr[1];
	iGrdpos_ptr[1]=loc_ptr[1]+iy;
	
	//	for(Int ix=-scaledSupport_ptr[0]; ix <= scaledSupport_ptr[0]; ix++) 
	for(Int ix=supBLC_ptr[0]; ix <= supTRC_ptr[0]; ix++) 
	  {
	    iloc_ptr[0]=(Int)((scaledSampling_ptr[0]*ix+off_ptr[0])-1)+convOrigin_ptr[0];
	    iGrdpos_ptr[0]=loc_ptr[0]+ix;
	    {
	      if (ix==0 and iy==0) foundCFPeak=True;
	      wt = getFrom4DArray((const Complex * __restrict__ &)convFuncV, 
				  iloc_ptr,cfInc_p)/cfArea;
	      if (wVal > 0.0) {wt = conj(wt);}
	      norm += (wt);
	      if (finitePointingOffset && !doPSFOnly) 
		wt *= cached_phaseGrad_p[iloc_ptr[0]+phaseGradOrigin_l[0]+
					 iloc_ptr[1]+phaseGradOrigin_l[1]*cachedPhaseGradNY];
	      
	      // The following uses raw index on the 4D grid
	      addTo4DArray(gridStore,iGrdpos_ptr,gridInc_p, nvalue,wt);
	    }
	  }
      }
    return norm;
  }
#include "cDataToGridImpl_def.h"
  
};
