#include<hip/hip_runtime.h>
#include <complex>
#include "/usr/local/cuda-5.5/include/hipfft/hipfft.h"

namespace casa 
{
    //CUFFT Call replacing the FFT call in AntenaaAterm.cc file

    int call_cufft(hipfftComplex *pointer, int  NX, int NY)
    {
        printf("Inside Call_cuda.cu file\n");
        hipfftHandle plan;

        printf("sizeof(hipfftComplex) = %d NX=%d NY=%d\n", sizeof(hipfftComplex), NX, NY);
        hipMalloc((void**)&pointer, sizeof(hipfftComplex)*NX*(NY));
        if (hipGetLastError() != hipSuccess){
            fprintf(stderr, "Cuda error: Failed to allocate\n");
            return 0;
        }

        /* Create a 2D FFT plan. */
        if (hipfftPlan2d(&plan, NX, NY, HIPFFT_C2C) != HIPFFT_SUCCESS){
            fprintf(stderr, "CUFFT Error: Unable to create plan\n");
            return 0;
        }


        if (cufftSetCompatibilityMode(plan, CUFFT_COMPATIBILITY_NATIVE)!= HIPFFT_SUCCESS){
            fprintf(stderr, "CUFFT Error: Unable to set compatibility mode to native\n");
            return 0;
        }

        if (hipfftExecC2C(plan, (hipfftComplex *)pointer, (hipfftComplex *)pointer, HIPFFT_FORWARD) != HIPFFT_SUCCESS){
            fprintf(stderr, "CUFFT Error: Unable to execute plan\n");
            return 0;
        }
        if (hipDeviceSynchronize() != hipSuccess){
  	    fprintf(stderr, "Cuda error: Failed to synchronize\n");
   	    return 0;
        }
        printf("After devicesync\n");
#if 1
        hipfftDestroy(plan);
        hipFree(pointer);
#endif

        return 0;
    }
}
