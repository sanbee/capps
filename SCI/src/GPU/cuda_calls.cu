#include<hip/hip_runtime.h>
#include <complex>
#include "/usr/local/cuda-5.5/include/hipfft/hipfft.h"

namespace casa 
{
    //CUFFT Call replacing the FFT call in AntenaaAterm.cc file

    int call_cufft(hipfftDoubleComplex *pointer, int  NX, int NY)
    {
        printf("Inside Call_cuda.cu file\n");
        hipfftHandle plan;

        hipMalloc((void**)&pointer, sizeof(hipfftDoubleComplex)*NX*(NY));
        if (hipGetLastError() != hipSuccess){
            fprintf(stderr, "Cuda error: Failed to allocate\n");
            return 0;
        }
        printf("cudamalloc done\n");

        /* Create a 2D FFT plan. */
        if (hipfftPlan2d(&plan, NX, NY, HIPFFT_C2C) != HIPFFT_SUCCESS){
            fprintf(stderr, "CUFFT Error: Unable to create plan\n");
            return 0;
        }
        printf("2d FFT Plan is done\n");

        if (cufftSetCompatibilityMode(plan, CUFFT_COMPATIBILITY_NATIVE)!= HIPFFT_SUCCESS){
            fprintf(stderr, "CUFFT Error: Unable to set compatibility mode to native\n");
            return 0;
        }
        printf("After comtmode\n");

        if (hipfftExecC2C(plan, (hipfftComplex *)pointer, (hipfftComplex *)pointer, HIPFFT_FORWARD) != HIPFFT_SUCCESS){
            fprintf(stderr, "CUFFT Error: Unable to execute plan\n");
            return 0;
        }
        printf("After cufftexec\n");
        return 0;
    }
}
