#include<hip/hip_runtime.h>
#include <complex>
#include <complex.h>
#include "/usr/local/cuda-5.5/include/hipfft/hipfft.h"
#include "AntennaATerm.h"

namespace casa 
{
    //CUFFT Call replacing the FFT call in AntenaaAterm.cc file

    int call_cufft(Complex *h_pointer, int  NX, int NY)
    {
        printf("Inside Call_cuda.cu file\n");
        hipfftHandle plan;
        hipfftComplex *d_pointer;

        printf("sizeof(hipfftComplex) = %d NX=%d NY=%d\n", sizeof(hipfftComplex), NX, NY);

        #if 0 
        int i=0;

        Complex tmp=0.0;
        for(i=0;i<NX*NY ; i+=100)
        {
             if (h_pointer[i] > tmp) tmp=h_pointer[i];
        }
        cout << "Max in cuda_calls= " << tmp << endl;
        #endif


        hipMalloc((void**)&d_pointer, sizeof(hipfftComplex)*NX*(NY));
        if (hipGetLastError() != hipSuccess){
            fprintf(stderr, "Cuda error: Failed to allocate\n");
            return 0;
        }
       
        hipMemcpy(d_pointer, h_pointer, sizeof(hipfftComplex)*NX*(NY), hipMemcpyHostToDevice);
        if (hipGetLastError() != hipSuccess){
            fprintf(stderr, "Cuda error: Failed to allocate\n");
            return 0;
        }
        

        /* Create a 2D FFT plan. */
        if (hipfftPlan2d(&plan, NX, NY, HIPFFT_C2C) != HIPFFT_SUCCESS){
            fprintf(stderr, "CUFFT Error: Unable to create plan\n");
            return 0;
        }



        //if (cufftSetCompatibilityMode(plan, CUFFT_COMPATIBILITY_NATIVE)!= HIPFFT_SUCCESS){
        if (cufftSetCompatibilityMode(plan, CUFFT_COMPATIBILITY_FFTW_PADDING)!= HIPFFT_SUCCESS){
            fprintf(stderr, "CUFFT Error: Unable to set compatibility mode to native\n");
            return 0;
        }

        if (hipfftExecC2C(plan, d_pointer, d_pointer, HIPFFT_FORWARD) != HIPFFT_SUCCESS){
            fprintf(stderr, "CUFFT Error: Unable to execute plan\n");
            return 0;
        }
        hipMemcpy(h_pointer, d_pointer, sizeof(hipfftComplex)*NX*(NY), hipMemcpyDeviceToHost);
        if (hipGetLastError() != hipSuccess){
            fprintf(stderr, "Cuda error: Failed to allocate\n");
            return 0;
        }
       
        printf("After devicesync\n");
#if 1
        hipfftDestroy(plan);
        hipFree(d_pointer);
#endif
#if 0
        int p;
        for (p=0;p<NX*NY;p++)
        cout << "FFTed Data by GPU = " << h_pointer[p] << endl;
        #endif

        return 0;
    }
}
