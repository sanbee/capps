#include "hip/hip_runtime.h"
// -*- C -*-
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <Utils.h>
#include <math.h>
#include <cuUtils.h>

#define USE_AUTO 
#undef USE_AUTO 
#define GRIDSIZE 2048
#define BLOCKSIZE 32

namespace casa{
  //
  //--------------------------------------------
  //
  void * allocateDeviceBuffer(int N)
  {
    void *d_buf;
    hipMalloc((void**)&d_buf, N);
    if (hipGetLastError() != hipSuccess)
      {
	fprintf(stderr, "Cuda error: Failed to allocate\n");
	return 0;
      }
    
    return d_buf;
  }
  //
  //--------------------------------------------
  //
  int sendBufferToDevice(void *d_buf, void *h_buf, int N)
  {
    hipMemcpy(d_buf, h_buf, N, hipMemcpyHostToDevice);
    if (hipGetLastError() != hipSuccess)
      {
	fprintf(stderr, "Cuda error: Failed to send\n");
	return 0;
      }
    return 1;
  }
  //
  //--------------------------------------------
  //
  int getBufferFromDevice(void *h_buf, void *d_buf, int N)
  {
    hipMemcpy(h_buf, d_buf, N, hipMemcpyDeviceToHost);
    if (hipGetLastError() != hipSuccess)
      {
	fprintf(stderr, "Cuda error: Failed to get\n");
	return 0;
      }
    return 1;
  }
  //
  //--------------------------------------------
  //
  int makeCUFFTPlan(hipfftHandle *plan, int NX, int NY, hipfftType type /*HIPFFT_C2C*/)
  {
    /* Create a 2D FFT plan. */
    if (hipfftPlan2d(plan, NX, NY, type) != HIPFFT_SUCCESS)
      {
	fprintf(stderr, "CUFFT Error: Unable to create plan\n");
	return 0;
      }
    return 1;
  }
  //
  //--------------------------------------------
  //
  int setCompatibilityMode(hipfftHandle& plan, cufftCompatibility mode /*CUFFT_COMPATIBILITY_NATIVE*/)
  {
    if (cufftSetCompatibilityMode(plan, mode)!= HIPFFT_SUCCESS)
      {
	fprintf(stderr, "CUFFT Error: Unable to set compatibility mode to native\n");
	return 0;
      }
    return 1;
  }
  //
  //--------------------------------------------
  //
  int inPlaceCUFFTC2C(hipfftHandle& plan, hipfftComplex *d_buf, int dir /*HIPFFT_FORWARD */)
  {
    if (hipfftExecC2C(plan, d_buf, d_buf, dir) != HIPFFT_SUCCESS)
      {
	fprintf(stderr, "CUFFT Error: Unable to execute plan\n");
	return 0;
      }
    hipDeviceSynchronize();
    return 1;
  }
  //
  //--------------------------------------------
  //
  hipError_t
    freeHost(void* h_mem, memoryMode memMode)
  {
    if( PINNED == memMode ) {
      return hipHostFree(h_mem);
    }
    else {
      free(h_mem);
    }
    return hipSuccess;
  }
  //
  //--------------------------------------------
  //
  hipError_t
    mallocHost(void** h_mem ,uint memSize, memoryMode memMode, bool wc)
  {
    if( PINNED == memMode ) {
#if CUDART_VERSION >= 2020
      return hipHostAlloc( h_mem, memSize, (wc) ? hipHostMallocWriteCombined : 0 );
#else
      if (wc) {printf("Write-Combined unavailable on CUDART_VERSION less than 2020, running is: %d", CUDART_VERSION);
        return hipHostMalloc( h_mem, memSize );
#endif
      }
      else { // PAGEABLE memory mode
        *h_mem = malloc( memSize );
      }
      
      return hipSuccess;
    }
    //
    //--------------------------------------------
    //
    hipError_t
      memCpy(void* sink, void* source, uint memSize, hipMemcpyKind direction, memoryMode memMode)
    {
      /* if( PINNED == memMode ) { */
      /*   return hipMemcpyAsync( sink, source, memSize, direction, 0); */
      /* } */
      /* else { */
      /*   return hipMemcpy( sink, source, memSize, direction); */
      /* } */
      return hipMemcpy( sink, source, memSize, direction);
    }
    //
    //--------------------------------------------
    //
    void cpu_wTermApplySky(hipfftComplex* screen, const int nx, const int ny,
			   const int TILE_WIDTH, const double wPixel,
			   const float sampling, const double wScale, 
			   const int inner,      const bool isNoOp)
    {
      double wValue=(wPixel*wPixel)/wScale;
      double twoPiW=2.0*M_PI*double(wValue);
      int convSize = nx;
      
      if (!isNoOp)
      	{
      	  for (int iy=-inner/2;iy<inner/2;iy++)
      	    {
      	      double m=sampling*double(iy);
      	      double msq=m*m;
      	      for (int ix=-inner/2;ix<inner/2;ix++)
      		{
      		  double l=sampling*double(ix);
      		  double rsq=l*l+msq;
      		  if(rsq<1.0)
      		    {
      		      double phase=twoPiW*(sqrt(1.0-rsq)-1.0);
		      hipfftComplex w;w.x=cos(phase); w.y=sin(phase);
		      screen[ix+convSize/2 + (iy+convSize/2)*ny]=
			hipCmulf(screen[ix+convSize/2 + (iy+convSize/2)*ny], w); 
      		      /* float wre=cos(phase), wim=sin(phase); */
      		      /* float re=screen[ix+convSize/2 + (iy+convSize/2)*ny].x, */
      		      /* 	im=screen[ix+convSize/2 + (iy+convSize/2)*ny].y; */
      		      /* screen[ix+convSize/2 + (iy+convSize/2)*ny].x=re*wre - im*wim; */
      		      /* screen[ix+convSize/2 + (iy+convSize/2)*ny].y=re*wim + im*wre; */
      		    }
      		}
      	    }
      	}
    }
    //
    //===========================================
    //--------------------------------------------
    //
    __global__ void kernel_wTermApplySky(hipfftComplex* screen, const int nx, const int ny,
					 const int TILE_WIDTH, const double wPixel,
					 const float sampling, const double wScale, 
					 const int inner,      const bool isNoOp)
    {
      int WIDTH=ny;
      
      unsigned int col = TILE_WIDTH*blockIdx.x + threadIdx.x ;
      unsigned int row = TILE_WIDTH*blockIdx.y + threadIdx.y ;
      double wValue=(wPixel*wPixel)/wScale;
      int convSize = nx;
      
      double twoPiW=2.0*M_PI*double(wValue);
      
      int ix=row-inner/2, iy=col-inner/2;
      double m=sampling*double(ix), l=sampling*double(iy);
      double rsq=(l*l+m*m);
      
      if (rsq<1.0)
	{
	  double phase=twoPiW*(sqrt(1.0-rsq)-1.0);
	  int tix=ix+convSize/2, tiy=iy+convSize/2;
	  hipfftComplex w;w.x=cos(phase); w.y=sin(phase);
	  
	  /* float wre=cos(phase), wim=sin(phase); */
	  /* float re=screen[row*WIDTH+col].x, */
	  /*   im=screen[row*WIDTH+col].y; */
	  /* screen[tix*WIDTH+tiy].x=re*wre - im*wim; */
	  /* screen[tix*WIDTH+tiy].y=re*wim + im*wre; */
	  
	  screen[tix*WIDTH+tiy] = hipCmulf(screen[tix*WIDTH+tiy], w);
	}
      
      
      /* if (!isNoOp) */
      /* 	{ */
      /* 	  for (int iy=-inner/2;iy<inner/2;iy++)  */
      /* 	    { */
      /* 	      double m=sampling*double(iy); */
      /* 	      double msq=m*m; */
      /* 	      for (int ix=-inner/2;ix<inner/2;ix++)  */
      /* 		{ */
      /* 		  double l=sampling*double(ix); */
      /* 		  double rsq=l*l+msq; */
      /* 		  if(rsq<1.0)  */
      /* 		    { */
      /* 		      double phase=twoPiW*(sqrt(1.0-rsq)-1.0); */
      /* 		      float re=screen[ix+convSize/2 + (iy+convSize/2)*ny].x, */
      /* 			im=screen[ix+convSize/2 + (iy+convSize/2)*ny].y; */
      /* 		      float wre=cos(phase), wim=sin(phase); */
      /* 		      screen[ix+convSize/2 + (iy+convSize/2)*ny].x=re*wre - im*wim; */
      /* 		      screen[ix+convSize/2 + (iy+convSize/2)*ny].y=re*wim + im*wre; */
      /* 		    } */
      /* 		} */
      /* 	    } */
      /* 	} */
    }
    //
    //--------------------------------------------
    //
    void wTermApplySky(hipfftComplex* screen,  const int& nx, const int& ny,
		       const int& TILE_WIDTH, const double& wPixel,
		       const float& sampling, const double& wScale, 
		       const int& inner,      const bool& isNoOp)
    {
#ifdef USE_AUTO
      {
	int WIDTH=ny;
	dim3 dimGrid ( WIDTH/TILE_WIDTH , WIDTH/TILE_WIDTH ,1 ) ;
	dim3 dimBlock( TILE_WIDTH, TILE_WIDTH, 1 ) ;
	
	kernel_wTermApplySky <<<dimGrid,dimBlock>>> (screen, nx, ny, TILE_WIDTH,wPixel, sampling,
						     wScale, inner,isNoOp);
      }
#else
      {
	dim3 dimGrid ( GRIDSIZE , 1 ,1 ) ;
	dim3 dimBlock( BLOCKSIZE,1,1);
	kernel_wTermApplySky <<<dimGrid,dimBlock>>> (screen, nx, ny, TILE_WIDTH,wPixel, sampling, 
						     wScale, inner,isNoOp);
      }
#endif
    }
    //
    //===========================================
    //--------------------------------------------
    //
    __global__ void kernel_setBuf(hipfftComplex *d_buf, const int nx, const int ny, 
				  const int TILE_WIDTH, hipfftComplex val)
    {
      int WIDTH=ny;
      
      // calculate thread id
      unsigned int col = TILE_WIDTH*blockIdx.x + threadIdx.x ;
      unsigned int row = TILE_WIDTH*blockIdx.y + threadIdx.y ;
      d_buf[row*WIDTH+col] = val;
    }
    //
    //--------------------------------------------
    //
    void setBuf(hipfftComplex *d_buf, const int nx, const int ny, 
		const int TILE_WIDTH, hipfftComplex val)
    {
#ifdef USE_AUTO
      {
	int WIDTH=ny;
	dim3 dimGrid ( WIDTH/TILE_WIDTH , WIDTH/TILE_WIDTH ,1 ) ;
	dim3 dimBlock( TILE_WIDTH, TILE_WIDTH, 1 ) ;
	
	kernel_setBuf<<<dimGrid,dimBlock>>> ( d_buf,nx,ny,TILE_WIDTH,val);
      }
#else
      {
	dim3 dimGrid ( GRIDSIZE ,1 ,1 ) ;
	dim3 dimBlock( BLOCKSIZE, 1, 1 ) ;
	
	kernel_setBuf<<<dimGrid,dimBlock>>> ( d_buf,nx,ny,TILE_WIDTH,val);
      }
#endif
    }
    //
    //===========================================
    //--------------------------------------------
    //
    __global__ void kernel_mulBuf(hipfftComplex *target_d_buf, const hipfftComplex* source_d_buf, 
				  const int nx, const int ny, const int TILE_WIDTH)
    {
      int WIDTH=ny;
      
      // calculate thread id
      unsigned int col = TILE_WIDTH*blockIdx.x + threadIdx.x ;
      unsigned int row = TILE_WIDTH*blockIdx.y + threadIdx.y ;
      target_d_buf[row*WIDTH+col] = hipCmulf(target_d_buf[row*WIDTH+col], source_d_buf[row*WIDTH+col]);
    }
    //
    //--------------------------------------------
    //
    void mulBuf(hipfftComplex *target_d_buf, const hipfftComplex* source_d_buf, 
		const int& nx, const int& ny, const int TILE_WIDTH)
    {
#ifdef USE_AUTO
      {
	int WIDTH=ny;
	dim3 dimGrid ( WIDTH/TILE_WIDTH , WIDTH/TILE_WIDTH ,1 ) ;
	dim3 dimBlock( TILE_WIDTH, TILE_WIDTH, 1 ) ;
	
	kernel_mulBuf<<<dimGrid,dimBlock>>>(target_d_buf, source_d_buf, nx,ny,TILE_WIDTH);
      }
#else
      {
	dim3 dimGrid ( GRIDSIZE, 1 ,1 ) ;
	dim3 dimBlock( BLOCKSIZE, 1, 1 ) ;
	
	kernel_mulBuf<<<dimGrid,dimBlock>>>(target_d_buf, source_d_buf, nx,ny,TILE_WIDTH);
      }
#endif
    }
    //
    //--------------------------------------------
    //
    void cpuflip(hipfftComplex *buf, const int nx, const int ny, const int TILE_WIDTH)
    {
      int cx=nx/2, cy=ny/2;
      
      for (int i=0; i<cx; i++)
	for (int j=0; j< cy; j++)
	  {
	    hipfftComplex tmp;
	    tmp=buf[i+j*ny];
	    buf[i+j*ny] = buf[cx+i + (cy+j)*ny];
	    buf[cx+i + (cy+j)*ny] = tmp;
	  }
      for (int i=cx; i < nx; i++)
	for (int j=0; j < cy; j++)
	  {
	    hipfftComplex tmp;
	    tmp=buf[i-cx +(j+cy)*ny];
	    buf[i-cx +(j+cy)*ny] = buf[i + j*ny];
	    buf[i + j*ny] = tmp;
	  }
    }
    //
    //===========================================
    // Following is the GPU kernel equivalent of the cpuflip function
    //
    __global__ void kernel_flip(hipfftComplex *buf, const int nx, const int ny, const int TILE_WIDTH)
    {
      // calculate thread id
      unsigned int i = TILE_WIDTH*blockIdx.x + threadIdx.x ;
      unsigned int j = TILE_WIDTH*blockIdx.y + threadIdx.y ;
      
      int cx=nx/2, cy=ny/2;
      hipfftComplex tmp;
      
      if (i < cx)
	{
	  tmp=buf[i+j*ny];
	  buf[i+j*ny] = buf[cx+i + (cy+j)*ny];
	  buf[cx+i + (cy+j)*ny] = tmp;
	}
      else
	{
	  tmp=buf[i-cx +(j+cy)*ny];
	  buf[i-cx +(j+cy)*ny] = buf[i + j*ny];
	  buf[i + j*ny] = tmp;
	}
    }
    //
    //--------------------------------------------
    //
    void flip(hipfftComplex *buf, const int nx, const int ny, const int TILE_WIDTH)
    {
#ifdef USE_AUTO
      {
	dim3 dimGrid ( nx/TILE_WIDTH , ny/(2*TILE_WIDTH) ,1 ) ;
	dim3 dimBlock( TILE_WIDTH, TILE_WIDTH, 1 ) ;
	
	kernel_flip<<<dimGrid,dimBlock>>>(buf, nx,ny,TILE_WIDTH);
      }
#else
      {
	dim3 dimGrid ( GRIDSIZE , 1 ,1 ) ;
	dim3 dimBlock( BLOCKSIZE, 1, 1 ) ;
	
	kernel_flip<<<dimGrid,dimBlock>>>(buf, nx,ny,TILE_WIDTH);
      }
#endif
    }
    //
    //============================================
    //--------------------------------------------
    //
    __global__ void kernel_flipSign(hipfftComplex *buf, const int nx, const int ny, const int TILE_WIDTH)
    {
      // calculate thread id
      unsigned int i = TILE_WIDTH*blockIdx.x + threadIdx.x ;
      unsigned int j = TILE_WIDTH*blockIdx.y + threadIdx.y ;
      
      /* for (int i=0; i<nx; i++) */
      /* 	for (int j=0; j<ny; j++) */
      {
	float sign=powf(-1.0,i+j);
	buf[i + j*ny].x = buf[i + j*ny].x*sign;
	buf[i + j*ny].y = buf[i + j*ny].y*sign;
      }
    }
    //
    //--------------------------------------------
    //
    void flipSign(hipfftComplex *buf, const int nx, const int ny, const int TILE_WIDTH)
    {
#ifdef USE_AUTO
      {
	dim3 dimGrid ( nx/TILE_WIDTH , ny/TILE_WIDTH ,1 ) ;
	dim3 dimBlock( TILE_WIDTH, TILE_WIDTH, 1 ) ;
	kernel_flipSign<<<dimGrid,dimBlock>>>(buf, nx,ny,TILE_WIDTH);
      }
#else
      {
	dim3 dimGrid ( GRIDSIZE , 1 ,1 ) ;
	dim3 dimBlock( BLOCKSIZE, 1, 1 ) ;
	kernel_flipSign<<<dimGrid,dimBlock>>>(buf, nx,ny,TILE_WIDTH);
      }
#endif
    }
    
  };
