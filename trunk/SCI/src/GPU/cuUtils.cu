#include "hip/hip_runtime.h"
// -*- C -*-
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <Utils.h>
#include <math.h>
#include <cuUtils.h>

namespace casa{
  //
  //--------------------------------------------
  //
  void * allocateDeviceBuffer(int N)
  {
    void *d_buf;
    hipMalloc((void**)&d_buf, N);
    if (hipGetLastError() != hipSuccess)
      {
	fprintf(stderr, "Cuda error: Failed to allocate\n");
	return 0;
      }
    
    return d_buf;
  }
  //
  //--------------------------------------------
  //
  int sendBufferToDevice(void *d_buf, void *h_buf, int N)
  {
    hipMemcpy(d_buf, h_buf, N, hipMemcpyHostToDevice);
    if (hipGetLastError() != hipSuccess)
      {
	fprintf(stderr, "Cuda error: Failed to send\n");
	return 0;
      }
    return 1;
  }
  //
  //--------------------------------------------
  //
  int getBufferFromDevice(void *h_buf, void *d_buf, int N)
  {
    hipMemcpy(h_buf, d_buf, N, hipMemcpyDeviceToHost);
    if (hipGetLastError() != hipSuccess)
      {
	fprintf(stderr, "Cuda error: Failed to get\n");
	return 0;
      }
    return 1;
  }
  //
  //--------------------------------------------
  //
  int makeCUFFTPlan(hipfftHandle *plan, int NX, int NY, hipfftType type /*HIPFFT_C2C*/)
  {
    /* Create a 2D FFT plan. */
    if (hipfftPlan2d(plan, NX, NY, type) != HIPFFT_SUCCESS)
      {
	fprintf(stderr, "CUFFT Error: Unable to create plan\n");
	return 0;
      }
    return 1;
  }
  //
  //--------------------------------------------
  //
  int setCompatibilityMode(hipfftHandle& plan, cufftCompatibility mode /*CUFFT_COMPATIBILITY_NATIVE*/)
  {
    if (cufftSetCompatibilityMode(plan, mode)!= HIPFFT_SUCCESS)
      {
	fprintf(stderr, "CUFFT Error: Unable to set compatibility mode to native\n");
	return 0;
      }
    return 1;
  }
  //
  //--------------------------------------------
  //
  int inPlaceCUFFTC2C(hipfftHandle& plan, hipfftComplex *d_buf, int dir /*HIPFFT_FORWARD */)
  {
    if (hipfftExecC2C(plan, d_buf, d_buf, dir) != HIPFFT_SUCCESS)
      {
	fprintf(stderr, "CUFFT Error: Unable to execute plan\n");
	return 0;
      }
    hipDeviceSynchronize();
    return 1;
  }
  //
  //--------------------------------------------
  //
  hipError_t
    freeHost(void* h_mem, memoryMode memMode)
  {
    if( PINNED == memMode ) {
      return hipHostFree(h_mem);
    }
    else {
      free(h_mem);
    }
    return hipSuccess;
  }
  //
  //--------------------------------------------
  //
  hipError_t
    mallocHost(void** h_mem ,uint memSize, memoryMode memMode, bool wc)
  {
    if( PINNED == memMode ) {
#if CUDART_VERSION >= 2020
      return hipHostAlloc( h_mem, memSize, (wc) ? hipHostMallocWriteCombined : 0 );
#else
      if (wc) {printf("Write-Combined unavailable on CUDART_VERSION less than 2020, running is: %d", CUDART_VERSION);
        return hipHostMalloc( h_mem, memSize );
#endif
      }
      else { // PAGEABLE memory mode
        *h_mem = malloc( memSize );
      }
      
      return hipSuccess;
    }
  //
  //--------------------------------------------
  //
    hipError_t
      memCpy(void* sink, void* source, uint memSize, hipMemcpyKind direction, memoryMode memMode)
    {
      /* if( PINNED == memMode ) { */
      /*   return hipMemcpyAsync( sink, source, memSize, direction, 0); */
      /* } */
      /* else { */
      /*   return hipMemcpy( sink, source, memSize, direction); */
      /* } */
        return hipMemcpy( sink, source, memSize, direction);
    }
  //
  //--------------------------------------------
  //
    void cpu_wTermApplySky(hipfftComplex* screen, const int nx, const int ny,
			   const int TILE_WIDTH, const double wPixel,
			   const float sampling, const double wScale, 
			   const int inner,      const bool isNoOp)
    {
      double wValue=(wPixel*wPixel)/wScale;
      double twoPiW=2.0*M_PI*double(wValue);
      int convSize = nx;

      if (!isNoOp)
      	{
      	  for (int iy=-inner/2;iy<inner/2;iy++)
      	    {
      	      double m=sampling*double(iy);
      	      double msq=m*m;
      	      for (int ix=-inner/2;ix<inner/2;ix++)
      		{
      		  double l=sampling*double(ix);
      		  double rsq=l*l+msq;
      		  if(rsq<1.0)
      		    {
      		      double phase=twoPiW*(sqrt(1.0-rsq)-1.0);
		      hipfftComplex w;w.x=cos(phase); w.y=sin(phase);
		      screen[ix+convSize/2 + (iy+convSize/2)*ny]=
			hipCmulf(screen[ix+convSize/2 + (iy+convSize/2)*ny], w); 
      		      /* float wre=cos(phase), wim=sin(phase); */
      		      /* float re=screen[ix+convSize/2 + (iy+convSize/2)*ny].x, */
      		      /* 	im=screen[ix+convSize/2 + (iy+convSize/2)*ny].y; */
      		      /* screen[ix+convSize/2 + (iy+convSize/2)*ny].x=re*wre - im*wim; */
      		      /* screen[ix+convSize/2 + (iy+convSize/2)*ny].y=re*wim + im*wre; */
      		    }
      		}
      	    }
      	}
    }
  //
  //--------------------------------------------
  //
    __global__ void kernel_wTermApplySky(hipfftComplex* screen, const int nx, const int ny,
					 const int TILE_WIDTH, const double wPixel,
					 const float sampling, const double wScale, 
					 const int inner,      const bool isNoOp)
    {
      int WIDTH=ny;

      unsigned int col = TILE_WIDTH*blockIdx.x + threadIdx.x ;
      unsigned int row = TILE_WIDTH*blockIdx.y + threadIdx.y ;
      double wValue=(wPixel*wPixel)/wScale;
      int convSize = nx;

      double twoPiW=2.0*M_PI*double(wValue);
      
      int ix=row-inner/2, iy=col-inner/2;
      double m=sampling*double(ix);
      double l=sampling*double(iy);
      double rsq=(l*l+m*m);
      if (rsq<1.0)
	{
	  double phase=twoPiW*(sqrt(1.0-rsq)-1.0);
	  int tix=ix+convSize/2, tiy=iy+convSize/2;
	  hipfftComplex w;w.x=cos(phase); w.y=sin(phase);

	  /* float wre=cos(phase), wim=sin(phase); */
	  /* float re=screen[row*WIDTH+col].x, */
	  /*   im=screen[row*WIDTH+col].y; */
	  /* screen[tix*WIDTH+tiy].x=re*wre - im*wim; */
	  /* screen[tix*WIDTH+tiy].y=re*wim + im*wre; */

	  screen[tix*WIDTH+tiy] = hipCmulf(screen[tix*WIDTH+tiy], w);
	}


      /* if (!isNoOp) */
      /* 	{ */
      /* 	  for (int iy=-inner/2;iy<inner/2;iy++)  */
      /* 	    { */
      /* 	      double m=sampling*double(iy); */
      /* 	      double msq=m*m; */
      /* 	      for (int ix=-inner/2;ix<inner/2;ix++)  */
      /* 		{ */
      /* 		  double l=sampling*double(ix); */
      /* 		  double rsq=l*l+msq; */
      /* 		  if(rsq<1.0)  */
      /* 		    { */
      /* 		      double phase=twoPiW*(sqrt(1.0-rsq)-1.0); */
      /* 		      float re=screen[ix+convSize/2 + (iy+convSize/2)*ny].x, */
      /* 			im=screen[ix+convSize/2 + (iy+convSize/2)*ny].y; */
      /* 		      float wre=cos(phase), wim=sin(phase); */
      /* 		      screen[ix+convSize/2 + (iy+convSize/2)*ny].x=re*wre - im*wim; */
      /* 		      screen[ix+convSize/2 + (iy+convSize/2)*ny].y=re*wim + im*wre; */
      /* 		    } */
      /* 		} */
      /* 	    } */
      /* 	} */
    }
  //
  //--------------------------------------------
  //
    void wTermApplySky(hipfftComplex* screen,  const int& nx, const int& ny,
		       const int& TILE_WIDTH, const double& wPixel,
		       const float& sampling, const double& wScale, 
		       const int& inner,      const bool& isNoOp)
    {
      int WIDTH=ny;
      dim3 dimGrid ( WIDTH/TILE_WIDTH , WIDTH/TILE_WIDTH ,1 ) ;
      dim3 dimBlock( TILE_WIDTH, TILE_WIDTH, 1 ) ;

     kernel_wTermApplySky <<<dimGrid,dimBlock>>> (screen, nx, ny, TILE_WIDTH,wPixel, sampling, 
			   wScale, inner,isNoOp);
    }
  //
  //--------------------------------------------
  //
    __global__ void kernel_setBuf(hipfftComplex *d_buf, const int nx, const int ny, 
				  const int TILE_WIDTH, hipfftComplex val)
    {
      int WIDTH=ny;
      
      // calculate thread id
      unsigned int col = TILE_WIDTH*blockIdx.x + threadIdx.x ;
      unsigned int row = TILE_WIDTH*blockIdx.y + threadIdx.y ;
      d_buf[row*WIDTH+col] = val;
    }
  //
  //--------------------------------------------
  //
    void setBuf(hipfftComplex *d_buf, const int nx, const int ny, 
		const int TILE_WIDTH, hipfftComplex val)
    {
      int WIDTH=ny;
      dim3 dimGrid ( WIDTH/TILE_WIDTH , WIDTH/TILE_WIDTH ,1 ) ;
      dim3 dimBlock( TILE_WIDTH, TILE_WIDTH, 1 ) ;

      kernel_setBuf<<<dimGrid,dimBlock>>> ( d_buf,nx,ny,TILE_WIDTH,val);
    }
  //
  //--------------------------------------------
  //
    __global__ void kernel_mulBuf(hipfftComplex *target_d_buf, const hipfftComplex* source_d_buf, 
				  const int nx, const int ny, const int TILE_WIDTH)
    {
      int WIDTH=ny;
      
      // calculate thread id
      unsigned int col = TILE_WIDTH*blockIdx.x + threadIdx.x ;
      unsigned int row = TILE_WIDTH*blockIdx.y + threadIdx.y ;
      target_d_buf[row*WIDTH+col] = hipCmulf(target_d_buf[row*WIDTH+col], source_d_buf[row*WIDTH+col]);
    }
  //
  //--------------------------------------------
  //
    void mulBuf(hipfftComplex *target_d_buf, const hipfftComplex* source_d_buf, 
		const int& nx, const int& ny, const int TILE_WIDTH)
    {
      int WIDTH=ny;
      dim3 dimGrid ( WIDTH/TILE_WIDTH , WIDTH/TILE_WIDTH ,1 ) ;
      dim3 dimBlock( TILE_WIDTH, TILE_WIDTH, 1 ) ;

      kernel_mulBuf<<<dimGrid,dimBlock>>>(target_d_buf, source_d_buf, nx,ny,TILE_WIDTH);
    }
    
  };
