#include "hip/hip_runtime.h"
// -*- C -*-
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <Utils.h>
#include <math.h>
#include <cuUtils.h>
#include <vector>
/* #include <cuWTerm.h> */
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#define USE_AUTO 
//#undef USE_AUTO 
#define BLOCKSIZE 128
#define GRIDSIZE (2048*2048/128)

#define OVERSAMPLING 20

namespace casa{
  //
  //--------------------------------------------
  //
  void * allocateDeviceBuffer(int N)
  {
    void *d_buf;
    hipMalloc((void**)&d_buf, N);
    if (hipGetLastError() != hipSuccess)
      {
	fprintf(stderr, "Cuda error: Failed to allocate\n");
	return 0;
      }
    
    return d_buf;
  }
  //
  //--------------------------------------------
  //
  int sendBufferToDevice(void *d_buf, void *h_buf, int N)
  {
    hipMemcpy(d_buf, h_buf, N, hipMemcpyHostToDevice);
    if (hipGetLastError() != hipSuccess)
      {
	fprintf(stderr, "Cuda error: Failed to send\n");
	return 0;
      }
    return 1;
  }
  //
  //--------------------------------------------
  //
  int getBufferFromDevice(void *h_buf, void *d_buf, int N)
  {
    hipMemcpy(h_buf, d_buf, N, hipMemcpyDeviceToHost);
    if (hipGetLastError() != hipSuccess)
      {
	fprintf(stderr, "Cuda error: Failed to get\n");
	return 0;
      }
    return 1;
  }
  //
  //--------------------------------------------
  //
  int makeCUFFTPlan(hipfftHandle *plan, int NX, int NY, hipfftType type /*HIPFFT_C2C*/)
  {
    /* Create a 2D FFT plan. */
    if (hipfftPlan2d(plan, NX, NY, type) != HIPFFT_SUCCESS)
      {
	fprintf(stderr, "CUFFT Error: Unable to create plan\n");
	return 0;
      }
    return 1;
  }
  //
  //--------------------------------------------
  //
  int setCompatibilityMode(hipfftHandle& plan, cufftCompatibility mode /*CUFFT_COMPATIBILITY_NATIVE*/)
  {
    if (cufftSetCompatibilityMode(plan, mode)!= HIPFFT_SUCCESS)
      {
	fprintf(stderr, "CUFFT Error: Unable to set compatibility mode to native\n");
	return 0;
      }
    return 1;
  }
  //
  //--------------------------------------------
  //
  int inPlaceCUFFTC2C(hipfftHandle& plan, hipfftComplex *d_buf, int dir /*HIPFFT_FORWARD */)
  {
    if (hipfftExecC2C(plan, d_buf, d_buf, dir) != HIPFFT_SUCCESS)
      {
	fprintf(stderr, "CUFFT Error: Unable to execute plan\n");
	return 0;
      }
    hipDeviceSynchronize();
    return 1;
  }
  //
  //--------------------------------------------
  //
  hipError_t
    freeHost(void* h_mem, memoryMode memMode)
  {
    if( PINNED == memMode ) {
      return hipHostFree(h_mem);
    }
    else {
      free(h_mem);
    }
    return hipSuccess;
  }
  //
  //--------------------------------------------
  //
  hipError_t
    mallocHost(void** h_mem ,uint memSize, memoryMode memMode, bool wc)
  {
    if( PINNED == memMode ) {
#if CUDART_VERSION >= 2020
      return hipHostAlloc( h_mem, memSize, (wc) ? hipHostMallocWriteCombined : 0 );
#else
      if (wc) {printf("Write-Combined unavailable on CUDART_VERSION less than 2020, running is: %d", CUDART_VERSION);
        return hipHostMalloc( h_mem, memSize );
#endif
      }
      else { // PAGEABLE memory mode
        *h_mem = malloc( memSize );
      }
      
      return hipSuccess;
    }
    //
    //--------------------------------------------
    //
    hipError_t
      memCpy(void* sink, void* source, uint memSize, hipMemcpyKind direction, memoryMode memMode)
    {
      /* if( PINNED == memMode ) { */
      /*   return hipMemcpyAsync( sink, source, memSize, direction, 0); */
      /* } */
      /* else { */
      /*   return hipMemcpy( sink, source, memSize, direction); */
      /* } */
      return hipMemcpy( sink, source, memSize, direction);
    }
    //
    //--------------------------------------------
    //
    void cpu_wTermApplySky(hipfftComplex* screen, const int nx, const int ny,
			   const int tileWidthX, const int tileWidthY, 
			   const double wPixel,  const float sampling, 
			   const double wScale,  const int inner,      
			   const bool isNoOp)
    {
      double wValue=(wPixel*wPixel)/wScale;
      double twoPiW=2.0*M_PI*double(wValue);
      int convSize = nx;
      
      if (!isNoOp)
      	{
      	  for (int iy=-inner/2;iy<inner/2;iy++)
      	    {
      	      double m=sampling*double(iy);
      	      double msq=m*m;
      	      for (int ix=-inner/2;ix<inner/2;ix++)
      		{
      		  double l=sampling*double(ix);
      		  double rsq=l*l+msq;
      		  if(rsq<1.0)
      		    {
      		      double phase=twoPiW*(sqrt(1.0-rsq)-1.0);
		      hipfftComplex w;w.x=cos(phase); w.y=sin(phase);
		      screen[ix+convSize/2 + (iy+convSize/2)*ny]=
			hipCmulf(screen[ix+convSize/2 + (iy+convSize/2)*ny], w); 
      		      /* float wre=cos(phase), wim=sin(phase); */
      		      /* float re=screen[ix+convSize/2 + (iy+convSize/2)*ny].x, */
      		      /* 	im=screen[ix+convSize/2 + (iy+convSize/2)*ny].y; */
      		      /* screen[ix+convSize/2 + (iy+convSize/2)*ny].x=re*wre - im*wim; */
      		      /* screen[ix+convSize/2 + (iy+convSize/2)*ny].y=re*wim + im*wre; */
      		    }
      		}
      	    }
      	}
    }
    //
    //===========================================
    //--------------------------------------------
    //
    __global__ void kernel_wTermApplySky(hipfftComplex* screen, 
					 const hipfftComplex* aTerm, 
					 const int nx, const int ny,
					 const int tileWidthX, const int tileWidthY, 
					 const double wPixel,
					 const float sampling, const double wScale, 
					 const int inner,      const bool isNoOp)
    {
      unsigned int col = tileWidthX*blockIdx.x + threadIdx.x ;
      unsigned int row = tileWidthY*blockIdx.y + threadIdx.y ;

      __shared__ float twoPiW;
      twoPiW=__fmul_rn(2.0,M_PI);
      int originx=nx/2, originy=ny/2, tix, tiy;

      /* for (col=blockIdx.x * tileWidthX + threadIdx.x; col < nx; col +=tileWidthX * gridDim.x) */
      /* 	for (row=blockIdx.y * tileWidthY + threadIdx.y; row < ny; row +=tileWidthY * gridDim.y) */
	  {
      int ix=row-inner/2, iy=col-inner/2;
      tix=ix+originx; tiy=iy+originy;

      float m=sampling*float(ix), l=sampling*float(iy);
      float rsq=(l*l+m*m);
      /* float m=__fmul_rn(sampling,float(ix)), l=__fmul_rn(sampling,float(iy)); */
      /* float rsq=__fadd_rn(__fmul_rn(l,l),__fmul_rn(m,m)); */

      if (rsq<1.0)
	{
	  //	  float wValue = wPixel*wPixel/wScale;
	  float wValue=__fdividef((wPixel*wPixel),wScale);
	  twoPiW = __fmul_rn(twoPiW, wValue);
	  //	  float phase = twoPiW*(sqrt(1.0-rsq)-1.0);
	  float phase=__fmul_rn(twoPiW,
				(__fsqrt_rn(1.0-rsq)-1.0)
				);
	  hipfftComplex w; __sincosf(phase, &(w.y),&(w.x));
	  screen[tix*ny+tiy] = hipCmulf(w,aTerm[tix*ny+tiy]);
	  //screen[tix*ny+tiy] = w;
	}
      else
	{
	  screen[tix*ny+tiy] = make_hipFloatComplex(0.0,0.0);
	}
	  }
    }
    //
    //--------------------------------------------
    //
    void wTermApplySky(hipfftComplex* screen,  
		       const hipfftComplex* aTerm,  
		       const int& nx, const int& ny,
		       const int tileWidthX, const int tileWidthY, 
		       const double& wPixel,
		       const float& sampling, const double& wScale, 
		       const int& inner,      const bool& isNoOp)
    {
#ifdef USE_AUTO
      {
	int WIDTH=ny;
	dim3 dimGrid ( WIDTH/tileWidthX , WIDTH/tileWidthY ,1 ) ;
	dim3 dimBlock( tileWidthX, tileWidthY, 1 ) ;
	
	kernel_wTermApplySky <<<dimGrid,dimBlock>>> (screen, aTerm, nx, ny, tileWidthX, tileWidthY,wPixel, sampling,
						     wScale, inner,isNoOp);
      }
#else
      {
	dim3 dimGrid ( GRIDSIZE , 1 ,1 ) ;
	dim3 dimBlock( BLOCKSIZE,1,1);
	kernel_wTermApplySky <<<dimGrid,dimBlock>>> (screen, aTerm, nx, ny, tileWidthX, tileWidthY,wPixel, sampling, 
						     wScale, inner,isNoOp);
      }
#endif
    }
    //
    //===========================================
    //--------------------------------------------
    //
    __global__ void kernel_setBuf(hipfftComplex *d_buf, const int nx, const int ny, 
				  const int tileWidthX, const int tileWidthY, 
				  hipfftComplex val)
    {
      int WIDTH=ny;
      
      // calculate thread id
      unsigned int col = tileWidthX*blockIdx.x + threadIdx.x ;
      unsigned int row = tileWidthY*blockIdx.y + threadIdx.y ;
      d_buf[row*WIDTH+col] = val;
    }
    //
    //--------------------------------------------
    //
    void setBuf(hipfftComplex *d_buf, const int nx, const int ny, 
		const int tileWidthX, const int tileWidthY, 
		hipfftComplex val)
    {
#ifdef USE_AUTO
      {
	int WIDTH=ny;
	dim3 dimGrid ( WIDTH/tileWidthX , WIDTH/tileWidthY ,1 ) ;
	dim3 dimBlock( tileWidthX, tileWidthY, 1 ) ;
	
	kernel_setBuf<<<dimGrid,dimBlock>>> ( d_buf,nx,ny,tileWidthX, tileWidthY,val);
      }
#else
      {
	dim3 dimGrid ( GRIDSIZE ,1 ,1 ) ;
	dim3 dimBlock( BLOCKSIZE, 1, 1 ) ;
	
	kernel_setBuf<<<dimGrid,dimBlock>>> ( d_buf,nx,ny,tileWidthX, tileWidthY,val);
      }
#endif
    }
    //
    //===========================================
    //--------------------------------------------
    //
    __global__ void kernel_mulBuf(hipfftComplex *target_d_buf, const hipfftComplex* source_d_buf, 
				  const int nx, const int ny, const int tileWidthX, const int tileWidthY)
    {
      int WIDTH=ny;
      
      // calculate thread id
      unsigned int col = tileWidthX*blockIdx.x + threadIdx.x ;
      unsigned int row = tileWidthY*blockIdx.y + threadIdx.y ;
      target_d_buf[row*WIDTH+col] = hipCmulf(target_d_buf[row*WIDTH+col], source_d_buf[row*WIDTH+col]);
    }
    //
    //--------------------------------------------
    //
    void mulBuf(hipfftComplex *target_d_buf, const hipfftComplex* source_d_buf, 
		const int& nx, const int& ny, const int tileWidthX, const int tileWidthY)
    {
#ifdef USE_AUTO
      {
	int WIDTH=ny;
	dim3 dimGrid ( WIDTH/tileWidthX , WIDTH/tileWidthY ,1 ) ;
	dim3 dimBlock( tileWidthX, tileWidthY, 1 ) ;
	
	kernel_mulBuf<<<dimGrid,dimBlock>>>(target_d_buf, source_d_buf, nx,ny,tileWidthX, tileWidthY);
      }
#else
      {
	dim3 dimGrid ( GRIDSIZE, 1 ,1 ) ;
	dim3 dimBlock( BLOCKSIZE, 1, 1 ) ;
	
	kernel_mulBuf<<<dimGrid,dimBlock>>>(target_d_buf, source_d_buf, nx,ny,tileWidthX, tileWidthY);
      }
#endif
    }
    //
    //--------------------------------------------
    //
    void cpuflip(hipfftComplex *buf, const int nx, const int ny, const int tileWidthX, const int tileWidthY)
    {
      int cx=nx/2, cy=ny/2;
      //      hipComplex sign=make_hipFloatComplex(1.0,0.0);
      
      for (int i=0; i<cx; i++)
	for (int j=0; j< cy; j++)
	  {
	    hipfftComplex tmp;
	    // if (((cx+i)+(cy+j))%2 == 0) sign.x = -1.0; else sign.x = 1.0;
	    //tmp=hipCmulf(buf[i+j*ny],sign);
	    tmp=buf[i+j*ny];

	    //if (((i)+(j))%2 == 0) sign.x = -1.0;  else sign.x = 1.0;
	    //buf[i+j*ny] = hipCmulf(buf[cx+i + (cy+j)*ny],sign);
	    buf[i+j*ny] = buf[cx+i + (cy+j)*ny];

	    buf[cx+i + (cy+j)*ny] = tmp;
	  }
      for (int i=cx; i < nx; i++)
	for (int j=0; j < cy; j++)
	  {
	    hipfftComplex tmp;

	    /* if (((i-cx)+(j+cy))%2 == 0) sign.x = -1.0; else sign.x = 1.0; */
	    /* tmp=hipCmulf(buf[i-cx +(j+cy)*ny],sign); */
	    tmp=buf[i-cx +(j+cy)*ny];

	    /* if (((i)+(j))%2 == 0) sign.x = -1.0; else sign.x = 1.0; */
	    /* buf[i-cx +(j+cy)*ny] = hipCmulf(buf[i + j*ny],sign); */
	    buf[i-cx +(j+cy)*ny] = buf[i + j*ny];

	    buf[i + j*ny] = tmp;
	  }
    }
    //
    //===========================================
    // Following is the GPU kernel equivalent of the cpuflip function
    //
    //  +--------------------+
    //  |         :          |
    //  |         :          |
    //  |    1    :     2    |
    //  |         :          |
    //  |....................|
    //  |         :          |
    //  |    4    :     3    |
    //  |         :          |
    //  |         :          |
    //  +--------------------+
    //
    // This function copies data from quadrant 1 to 3 (and from 3 to 1) and 4 to 2 (and from 2 to 4).
    // While copying, it also now flips the sign of the pixel values if the target pixel (i,j) satisfies (i+j)%2 != 0.
    // This therefore effectively combines the flipSign() kernel in flip() kernel itself (i.e., flipSign() does not
    // need to be envoked).  This saves ~15% in run-time.
    //
#define FLIPSIGN(i,j,ny,val) ({if (((i)+(j))%2 != 0) {(val[(i)+(j)*(ny)]).x *=-1.0; (val[(i)+(j)*(ny)]).y *= -1.0;}})
    __global__ void kernel_flip(hipfftComplex *buf, const int nx, const int ny, const int tileWidthX, const int tileWidthY)
    {
      // calculate thread id
      unsigned int i = tileWidthX*blockIdx.x + threadIdx.x ;
      unsigned int j = tileWidthY*blockIdx.y + threadIdx.y ;
      unsigned int cx=nx/2, cy=ny/2;
      hipComplex sign=make_hipFloatComplex(1.0,0.0);

      hipfftComplex tmp;

      if (i < cx)
	{
	  if (((cx+i)+(cy+j))%2 == 0) sign.x=1.0; else sign.x = -1.0;
	  tmp=hipCmulf(buf[i+j*ny],sign);
	  
	  if ((i+j)%2 == 0) sign.x=1.0; else sign.x = -1.0;
	  buf[i+j*ny] = hipCmulf(buf[cx+i + (cy+j)*ny],sign);

	  buf[cx+i + (cy+j)*ny] = tmp;

	  /* The commented out code below is cleaner code, but which
	     runs ~10% slower!!!  Don't understand why. */

	  /* tmp=buf[i+j*ny]; */
	  /* buf[i+j*ny] = buf[cx+i + (cy+j)*ny]; */
	  /* FLIPSIGN(i,j,ny,buf); */
	  /* tx=cx+i; ty=cy+j; */
	  /* buf[tx + ty*ny] = tmp; */
	  /* FLIPSIGN(tx, ty,ny, buf); */
	}
      else
	{
	  if ((i+j)%2 == 0) sign.x=1.0; else sign.x = -1.0;
	  tmp=hipCmulf(buf[i-cx +(j+cy)*ny],sign);

	  if (((i-cx)+(j+cy))%2 == 0) sign.x=1.0; else sign.x = -1.0;
	  buf[i-cx +(j+cy)*ny] = hipCmulf(buf[i + j*ny],sign);
	    
	  buf[i + j*ny] = tmp;

	  /* The commented out code below is cleaner code, but which
	     runs ~10% slower!!!  Don't understand why. */

	  /* unsigned int tx,ty; */
	  /* tx=i-cx; ty=j+cy; */
	  /* tmp=buf[tx +ty*ny]; */
	  /* buf[i-cx +(j+cy)*ny] = buf[i + j*ny]; */
	  /* FLIPSIGN(tx, ty,ny, buf); */
	  /* buf[i + j*ny] = tmp; */
	  /* FLIPSIGN(i,j,ny,buf); */
	}
    }
//buf, 4,4,2,2
__global__ void kernel_newflip(hipfftComplex *buf, const int nx, const int ny, const int tileWidthX, const int tileWidthY)
    {
      // calculate thread id
      unsigned int i = tileWidthX*blockIdx.x + threadIdx.x ;
      unsigned int j = tileWidthY*blockIdx.y + threadIdx.y ;
      
      int cx=nx/2, cy=ny/2;
      hipfftComplex tmp;

      if (i < cx  && j <cy) 
        {
          //printf("i=%d, j=%d, tmp=%d, buf[%d]=%d\n", i,j, tmp, (cx+i + (cy+j)*ny), buf[cx+i + (cy+j)*ny]);
          tmp=buf[i+j*ny];
          buf[i+j*ny] = buf[cx+i + (cy+j)*ny];
          buf[cx+i + (cy+j)*ny] = tmp;
        }
      else if (j < cy)
        {
          //printf("i=%d, j=%d, cx=%d cy=%d nx=%d ny=%d\n",i,j,cx,cy,nx,ny);
          //printf("i=%d, j=%d, buf[%d]=%d buf_s[%d]=%d\n", i,j, (i-cx + (cy+j)*ny),buf[i-cx+(cy+j)*ny],(i + j*ny), buf[i + j*ny]);
          tmp=buf[i-cx +(j+cy)*ny];
          buf[i-cx +(j+cy)*ny] = buf[i + j*ny];
          buf[i + j*ny] = tmp;
        }
    }
    //
    //--------------------------------------------
    //
    //
    //--------------------------------------------
    //
    void flip(hipfftComplex *buf, const int nx, const int ny, const int tileWidthX, const int tileWidthY)
    {
      /* dim3 dimGrid ( (nx/tileWidthX) , (ny/tileWidthY) ,1 ) ; */
      /* dim3 dimBlock( tileWidthX, tileWidthY, 1 ) ; */

      /* kernel_newflip<<<dimGrid,dimBlock>>>(buf, nx,ny,tileWidthX, tileWidthY); */

#ifdef USE_AUTO
      {
	dim3 dimGrid ( nx/tileWidthX , ny/(2*tileWidthY) ,1 ) ;
	dim3 dimBlock( tileWidthX, tileWidthY, 1 ) ;
	
	kernel_flip<<<dimGrid,dimBlock>>>(buf, nx,ny,tileWidthX, tileWidthY);
      }
#else
      {
	dim3 dimGrid ( GRIDSIZE , 1 ,1 ) ;
	dim3 dimBlock( BLOCKSIZE, 1, 1 ) ;
	
	kernel_flip<<<dimGrid,dimBlock>>>(buf, nx,ny,tileWidthX, tileWidthY);
      }
#endif
    }
    //
    //============================================
    //--------------------------------------------
    //
    __global__ void kernel_flipSign(hipfftComplex *buf, const int nx, const int ny, const int tileWidthX, const int tileWidthY)
    {
      // calculate thread id
      unsigned int i = tileWidthX*blockIdx.x + threadIdx.x ;
      unsigned int j = tileWidthY*blockIdx.y + threadIdx.y ;
      float sign;
      {
	//	sign=__pow(-1.0,(float)(i+j));
	if ((i+j)%2 == 0) sign=1.0; else sign=-1.0;
	buf[i + j*ny].x = buf[i + j*ny].x*sign;
	buf[i + j*ny].y = buf[i + j*ny].y*sign;
      }
    }
    //
    //--------------------------------------------
    //
    void flipSign(hipfftComplex *buf, const int nx, const int ny, const int tileWidthX, const int tileWidthY)
    {
#ifdef USE_AUTO
      {
	dim3 dimGrid ( nx/tileWidthX , ny/tileWidthY ,1 ) ;
	dim3 dimBlock( tileWidthY, tileWidthY, 1 ) ;
	kernel_flipSign<<<dimGrid,dimBlock>>>(buf, nx,ny,tileWidthX, tileWidthY);
      }
#else
      {
	dim3 dimGrid ( GRIDSIZE , 1 ,1 ) ;
	dim3 dimBlock( BLOCKSIZE, 1, 1 ) ;
	kernel_flipSign<<<dimGrid,dimBlock>>>(buf, nx,ny,tileWidthX, tileWidthY);
      }
#endif
    }
    //
    //============================================
    //--------------------------------------------
    //
    //
    //============================================
    //--------------------------------------------
    //
    //
    //============================================
    //--------------------------------------------
    //
#undef HAS_OMP
#ifdef HAS_OMP
#include <omp.h>
#endif
    
    //
    //----------------------------------------------------------------------
    // A global method for use in OMP'ed findSupport() below
    //
    // void archPeak(const float& threshold, const int& origin, const Block<int>& cfShape, const Complex* funcPtr, 
    // 		const int& nCFS, const int& PixInc,const int& th, const int& R, 
    // 		Block<int>& maxR)
    __device__ void cuArchPeak(const float& threshold, const int& origin, const int nx, const int ny, 
			       const hipComplex* funcPtr, const int& PixInc,const int& th, 
			       const int& R, int* maxR)
    {
      /* thrust::device_vector<hipComplex> vals; */
      /* thrust::device_vector<int> ndx; ndx.assign(2,0); */
      int ndx[2];
      int NSteps;
      //Check every PixInc pixel along a circle of radius R
      NSteps = 90*R/PixInc; 
      int valsNelements=(int)(NSteps+0.5);
      //      vals.resize((int)(NSteps+0.5));
      hipComplex *vals;//[valsNelements];

      hipComplex zero=make_hipFloatComplex(0.0, 0.0);
      //vals.assign(valsNelements,zero);
      for (int ii=0;ii<valsNelements; ii++) vals[ii]=zero;
      //      vals=0;
      
      for(int pix=0;pix<NSteps;pix++)
	{
	  ndx[0]=(int)(origin + R*sin(2.0*M_PI*pix*PixInc/R));
	  ndx[1]=(int)(origin + R*cos(2.0*M_PI*pix*PixInc/R));
	  
	  if ((ndx[0] < nx) && (ndx[1] < ny))
	    //vals[pix]=func(ndx);
	    vals[pix]=funcPtr[ndx[0]+ndx[1]*nx];
	}
      
      maxR[th]=-R;
      for (uint i=0;i<valsNelements;i++)
	if (hipCabsf(vals[i]) > threshold)
	  {
	    maxR[th]=R;
	    break;
	  }
      //		th++;
    }

    __device__ bool cuFindSupport(hipComplex* funcPtr_d, const int nx, const int ny, 
				  float& threshold, int& origin, int& radius)
    {
      int PixInc=1, R0, R1, R, convSize;
      bool found=false;
      uint Nth=1, threadID=0;
      
      
      convSize = nx;
#ifdef HAS_OMP
      Nth = max(omp_get_max_threads()-2,1);
#endif
      
      int *maxR_p;
      /* std::vector<int> maxR(Nth); */
      /* maxR_p=maxR.data(); */
      
      R1 = convSize/2-2;
      radius=R1;
      while (R1 > 1)
	{
	  R0 = R1; R1 -= Nth;
	  
#pragma omp parallel default(none) firstprivate(R0,R1)  private(R,threadID) shared(origin, threshold, PixInc,maxR_p ,nCFS,funcPtr_d) num_threads(Nth)
	  { 
#pragma omp for
	    for(R=R0;R>R1;R--)
	      {
#ifdef HAS_OMP
		threadID=omp_get_thread_num();
#endif
		cuArchPeak(threshold, origin, nx, ny, funcPtr_d, PixInc, threadID, R, maxR_p);
	      }
	    ///#pragma omp barrier
	  }///omp 	    
	  
	  for (uint th=0;th<Nth;th++)
	    {
	      if (maxR_p[th] > 0)
		{
		  found=true; 
		  if (maxR_p[th] < radius) radius=maxR_p[th]; 
		}
	    }
	  if (found) 
	    return found;
	}
      return found;
    }
    
    __device__ bool cuSetUpCFSupport(hipComplex* func_d, const int nx, const int ny, 
			  int& xSupport, int& ySupport,
			  const float& sampling, const hipComplex& peak)
    {
      xSupport = ySupport = -1;
      //      int convFuncOrigin=func.shape()[0]/2, R; 
      int convFuncOrigin=nx/2, R; 
      bool found=false;
      float threshold;

      if (hipCabsf(peak) != 0) threshold = hipCabsf(peak);
      else 
	threshold   = (hipCabsf(func_d[convFuncOrigin+convFuncOrigin*ny]));

      threshold *= 1e-3;

      if (found = cuFindSupport(func_d,nx, ny,threshold,convFuncOrigin,R))
	xSupport=ySupport=int(0.5+float(R)/sampling)+1;

      if (xSupport*sampling > convFuncOrigin)
	  xSupport = ySupport = (int)(convFuncOrigin/sampling);

      return found;
    }
    
    __device__ bool cuResizeCF(hipComplex* func_d, const int nx, const int ny, 
		    int& xSupport, int& ySupport,
		    const float& sampling, const hipComplex& peak)
    {
      bool found = cuSetUpCFSupport(func_d, nx, ny, xSupport, ySupport, sampling,peak);
      
      return true;
      
      /* //int supportBuffer = aTerm_p->getOversampling()*2; */
      /* int ConvFuncOrigin=nx/2;  // Conv. Func. is half that size of convSize */
      
      /* int supportBuffer = OVERSAMPLING*2; */
      /* int bot=(int)(ConvFuncOrigin-sampling*xSupport-supportBuffer),//-convSampling/2,  */
      /* 	top=(int)(ConvFuncOrigin+sampling*xSupport+supportBuffer);//+convSampling/2; */
      /* //    bot *= 2; top *= 2; */
      /* bot = max(0,bot); */
      /* top = min(top, nx-1); */
      
      /* Array<Complex> tmp; */
      /* IPosition blc(4,bot,bot,0,0), trc(4,top,top,0,0); */
      /* // */
      /* // Cut out the conv. func., copy in a temp. array, resize the */
      /* // CFStore.data, and copy the cutout version to CFStore.data. */
      /* // */
      /* tmp = func(blc,trc); */
      /* func.resize(tmp.shape()); */
      /* func = tmp;  */
      /* return found; */
    }
    
  };
