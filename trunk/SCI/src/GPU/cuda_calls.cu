#include<hip/hip_runtime.h>

#include "/usr/local/cuda-5.5/include/hipfft/hipfft.h"
#if 0
//CUFFT Call replacing the FFT call in AntenaaAterm.cc file
int call_cufft(complex *pointer, int  NX, int NY)
{
    hipfftHandle plan;
    hipfftDoubleComplex *data;
    data = pointer;

    int n[NRANK] = {NX, NY};

    hipMalloc((void**)&data, sizeof(hipfftDoubleComplex)*NX*(NY));
    if (hipGetLastError() != hipSuccess){
        fprintf(stderr, "Cuda error: Failed to allocate\n");
        return;
    }

    /* Create a 2D FFT plan. */
    if (cufftPlan2D(&plan, NX, NY, HIPFFT_C2C) != HIPFFT_SUCCESS){
        fprintf(stderr, "CUFFT Error: Unable to create plan\n");
        return;
    }

    if (cufftSetCompatibilityMode(plan, CUFFT_COMPATIBILITY_NATIVE)!= HIPFFT_SUCCESS){
        fprintf(stderr, "CUFFT Error: Unable to set compatibility mode to native\n");
        return;
    }


    if (hipfftExecC2C(plan, data, data, HIPFFT_FORWARD) != HIPFFT_SUCCESS){
        fprintf(stderr, "CUFFT Error: Unable to execute plan\n");
        return;
    }
    return 0;
}
#endif
