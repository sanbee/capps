#include<hip/hip_runtime.h>
#include <complex>
#include <complex.h>
#include "/usr/local/cuda-5.5/include/hipfft/hipfft.h"
#include "AntennaATerm.h"

namespace casa 
{
    //CUFFT Call replacing the FFT call in AntenaaAterm.cc file

    int call_cufft(Complex *h_pointer, int  NX, int NY, int flag)
    {
        //if (flag == 1)
        //{
            hipfftHandle plan;
        //}

        hipfftComplex *d_pointer;


        hipMalloc((void**)&d_pointer, sizeof(hipfftComplex)*NX*(NY));
        if (hipGetLastError() != hipSuccess){
            fprintf(stderr, "Cuda error: Failed to allocate\n");
            return 0;
        }
       
        hipMemcpy(d_pointer, h_pointer, sizeof(hipfftComplex)*NX*(NY), hipMemcpyHostToDevice);
        if (hipGetLastError() != hipSuccess){
            fprintf(stderr, "Cuda error: Failed to allocate\n");
            return 0;
        }
        
        //if (flag == 1)
        {
            /* Create a 2D FFT plan. */
            if (hipfftPlan2d(&plan, NX, NY, HIPFFT_C2C) != HIPFFT_SUCCESS){
                fprintf(stderr, "CUFFT Error: Unable to create plan\n");
                return 0;
            }
        }



        if (cufftSetCompatibilityMode(plan, CUFFT_COMPATIBILITY_NATIVE)!= HIPFFT_SUCCESS){
            fprintf(stderr, "CUFFT Error: Unable to set compatibility mode to native\n");
            return 0;
        }

        if (hipfftExecC2C(plan, d_pointer, d_pointer, HIPFFT_FORWARD) != HIPFFT_SUCCESS){
            fprintf(stderr, "CUFFT Error: Unable to execute plan\n");
            return 0;
        }

        hipMemcpy(h_pointer, d_pointer, sizeof(hipfftComplex)*NX*(NY), hipMemcpyDeviceToHost);
        if (hipGetLastError() != hipSuccess){
            fprintf(stderr, "Cuda error: Failed to allocate\n");
            return 0;
        }
       
        //if (flag == 0)
        {
        hipfftDestroy(plan);
        }
        hipFree(d_pointer);

        return 0;
    }
}
