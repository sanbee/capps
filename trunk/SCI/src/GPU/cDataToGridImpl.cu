#include "hip/hip_runtime.h"
// -*- C++ -*-
//#include <synthesis/TransformMachines/cDataToGridImpl.h>
#include <casa/Arrays/Matrix.h>
//#include <hipfft/hipfft.h>
#include <hip/hip_complex.h>
#include "cDataToGridImpl.h"
#include <typeinfo>
#include <stdio.h>
#include "./GPUGEOM.h"

extern "C" {
#include <cuUtils.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime_api.h>
};

namespace casa{

#include "cDataToGridImpl_testcode.cu"

  template <class T>
  void cuDataToGridImpl_p(T* gridStore,  Int* gridShape, //4-elements
			  
  			  const uInt *subGridShape,//[2],
  			  const uInt *BLCXi, const uInt *BLCYi,
  			  const uInt *TRCXi, const uInt *TRCYi,
			  
  			  const Complex *visCube_ptr, const Float* imgWts_ptr,
  			  const Bool *flagCube_ptr, const Bool *rowFlag_ptr,
  			  const Double *uvw_ptr,
			  
  			  const Int nRow, const Int rbeg, const Int rend, 
  			  const Int nDataChan,const Int nDataPol, 
  			  const Int startChan, const Int endChan, const Int vbSpw,
  			  const Double *vbFreq,
			  
			  Complex **cfV, //[2]
  			  Int *cfShape,//[4], //[4]
  			  Float *sampling,//[2], 
			  const Int *support, //[2]
			  
  			  Double* sumWt_ptr,
  			  const Bool dopsf, const Bool accumCFs,
  			  const Int* polMap_ptr, const Int *chanMap_ptr,
  			  const Double *uvwScale_ptr, const Double *offset_ptr,
  			  const Double *dphase_ptr, Int XThGrid, Int YThGrid,
			  Int *gridHits)
  {
    // This is a host-side function!!!

    //    printf("DSubGridShape = %d %d\n", subGridShape[0], subGridShape[1]);
    // Int NB=16;
    // Int NT=17;
    Int NB=XBLOCKSIZE;
    Int NT=XTHREADSIZE;

    dim3 dimBlock ( NB, NB, 1 ) ;
    dim3 dimThread( NT, NT, 1 ) ;
    
    //hipProfilerStart();


    cuDataToGridImpl2_p<<<dimBlock,dimThread>>>(gridStore, gridShape,subGridShape,BLCXi,BLCYi,TRCXi,TRCYi,
						visCube_ptr,imgWts_ptr,flagCube_ptr,rowFlag_ptr,uvw_ptr,
						      
						nRow,rbeg,rend,nDataChan,nDataPol,startChan,endChan,vbSpw,
						vbFreq,
						      
						cfV,cfShape,sampling,support,
		       
						sumWt_ptr,dopsf,accumCFs,polMap_ptr,chanMap_ptr,
						uvwScale_ptr,offset_ptr,dphase_ptr,XThGrid,YThGrid,gridHits);



    //    hipDeviceSynchronize();

    // cDataToGridImpl2_p(gridStore, gridShape,subGridShape,BLCXi,BLCYi,TRCXi,TRCYi,
    // 		       visCube_ptr,imgWts_ptr,flagCube_ptr,rowFlag_ptr,uvw_ptr,
		       
    // 		       nRow,rbeg,rend,nDataChan,nDataPol,startChan,endChan,vbSpw,
    // 		       vbFreq,
		       
    // 		       cfV,cfShape,sampling,support,
		       
    // 		       sumWt_ptr,dopsf,accumCFs,polMap_ptr,chanMap_ptr,
    // 		       uvwScale_ptr,offset_ptr,dphase_ptr,XThGrid,YThGrid);
    
    hipError_t err=hipGetLastError();
    if (err != hipSuccess)
      {
	cerr << "###Cuda error: Failed to run the kernel " << hipGetErrorString (err) << endl;
	exit(0);
      }
    //hipProfilerStop();
  };
  //
  //---------------------------------------------------------------------------------
  // The following function is the CUDA kernel for gridding.
  //
  template <class T>
  __global__
  void cuDataToGridImpl2_p(T* gridStore,  Int* gridShape, //4-elements
			   
			   const uInt *subGridShape,//[2],
			   const uInt *BLCXi, const uInt *BLCYi,
			   const uInt *TRCXi, const uInt *TRCYi,
			   
			   const Complex *visCube_ptr, const Float* imgWts_ptr,
			   const Bool *flagCube_ptr, const Bool *rowFlag_ptr,
			   const Double *uvw_ptr,
			   
			   const Int nRow, const Int rbeg, const Int rend, 
			   const Int nDataChan,const Int nDataPol, 
			   const Int startChan, const Int endChan, const Int vbSpw,
			   const Double *vbFreq,
			   
			   Complex **cfV,//[2],
			   Int *cfShape,//[4], //[4]
			   Float *sampling,//[2], 
			   const Int *support, //[2]
			   
			   Double* sumWt_ptr,
			   const Bool dopsf, const Bool accumCFs,
			   const Int* polMap_ptr, const Int *chanMap_ptr,
			   const Double *uvwScale_ptr, const Double *offset_ptr,
			   const Double *dphase_ptr, Int XThGrid, Int YThGrid,
			   Int *gridHits)
  {
    XThGrid = blockIdx.x;
    YThGrid = blockIdx.y;

    Int nw, nCFFreq, nx,ny, nGridPol, nGridChan;
    Int targetIMChan, targetIMPol;
    
    Int loc[3], iloc[4],tiloc[4];
    Int convOrigin[4], gridInc_l[4], cfInc_l[4]; 
    Double pos[3], off[3];
    Int igrdpos[4];
    
    hipComplex phasor, nvalue;
    hipComplex norm;
    Bool Dummy;
    Double *pointingOffset_ptr=NULL;
    Double *cached_PointingOffset_ptr=NULL;
    
    nx=gridShape[0]; ny=gridShape[1];
    nGridPol=gridShape[2]; nGridChan=gridShape[3];
    Bool gDummy;

    CU_CACHE_AXIS_INCREMENTS(gridShape, gridInc_l);
    
    iloc[0]=iloc[1]=iloc[2]=iloc[3]=0;
    
    // Loop over all Rows, and all channels and polarization in each
    // row.  if the data[Row, Chan, Pol] data point is not flagged,
    // use it's UVW co-ordinate to determine if the current Block is
    // touched by the CF for this data point (this decision is in the
    // variable onMyGrid below).  
    //
    // If onMyGrid == True, then determine the pixels within this
    // block that need to do the addition.  With threads-per-block
    // equal to the support size of the CF (currently 8x8 pixels), the
    // CF pixels will map to the threadIdx.  This is done in the
    // cuaccumulateToGrid() function below.  Inside this function, we
    // loop over all the pixels of the CF (8x8 pixels), but do the
    // gridding (accumultion) only if the CF pixel index matches the
    // threadIdx.x and threadIdx.y.
    //

    convOrigin[0]=cfShape[0]/2;	    convOrigin[1]=cfShape[1]/2;
    convOrigin[2]=cfShape[2]/2;	    convOrigin[3]=cfShape[3]/2;
    CU_CACHE_AXIS_INCREMENTS(cfShape, cfInc_l);

    Bool finitePointingOffsets = False;
    for(Int irow=rbeg; irow< rend; irow++)
      {   
  	if(!(*(rowFlag_ptr+irow)))
  	  {   
	    const Float *imgWts_Chan_offset=imgWts_ptr + irow*nDataChan;
  	    for(Int ichan=startChan; ichan< endChan; ichan++)
  	      {
  		//if (*(imgWts_ptr + ichan+irow*nDataChan)!=0.0) 
		const Float imgWts_Chan = *(imgWts_Chan_offset+ichan);
		if ((imgWts_Chan)!=0.0) 
  		  {  
  		    targetIMChan=chanMap_ptr[ichan];
		    
  		    if((targetIMChan>=0) && (targetIMChan<nGridChan)) 
  		      {
  			Double dataWVal = 0;
  			// if (uvw_ptr != NULL) dataWVal = uvw_ptr[irow+nRow*2];
			
  			Int wndx = 0;//(int)(sqrt(vbs->cfBSt_p.wIncr*abs(dataWVal*vbFreq[ichan]/C::c)));
			
  			Int cfFreqNdx=0;
  			Float s;
			
  			cusgrid(pos,loc,off, &phasor, irow, uvw_ptr, dphase_ptr[irow], vbFreq[ichan], 
  			       uvwScale_ptr, offset_ptr, sampling);
			
  			Float cfblc[2], cftrc[2];

  			Bool onMyGrid=
			  cucomputeSupport(BLCXi,BLCYi, TRCXi, TRCYi, subGridShape,
					   XThGrid, YThGrid, support, sampling, pos, 
					   loc,cfblc,cftrc);

  			if (onMyGrid)
  			  {
			    // Gather some stats
			    // gridHits[XThGrid + YThGrid*subGridShape[0]]++;

			    //			    printf("%d %d %d\n",XThGrid, YThGrid,gridHits[XThGrid + YThGrid*subGridShape[0]]);
			    
			    Int iblc[2], itrc[2];
  			    iblc[0]=NINT((cfblc[0]-pos[0]));///sampling[0]);
  			    iblc[1]=NINT((cfblc[1]-pos[1]));///sampling[1]);
  			    itrc[0]=NINT((cftrc[0]-pos[0]));///sampling[0]);
  			    itrc[1]=NINT((cftrc[1]-pos[1]));///sampling[1]); 

  			    // Loop over all image-plane polarization planes.
  			    for(Int ipol=0; ipol< nDataPol; ipol++) 
  			      { 
				const Int iCiP_offset = ipol + ichan*nDataPol + irow*nDataPol*nDataChan;
				const Bool iCiPFlagCube = *(flagCube_ptr + iCiP_offset);
  				//if((!(*(flagCube_ptr + iCiP_offset))))
				if((!(iCiPFlagCube)))
  				  {  
  				    targetIMPol=polMap_ptr[ipol];
  				    if ((targetIMPol>=0) && (targetIMPol<nGridPol)) 
  				      {
  					igrdpos[2]=targetIMPol; igrdpos[3]=targetIMChan;
					
  					//if(dopsf) {nvalue.x=(*(imgWts_ptr + ichan + irow*nDataChan));nvalue.y=0.0;}
					if(dopsf) {nvalue.x=((imgWts_Chan));nvalue.y=0.0;}
  					else      
					  {
					    hipComplex vis;
					    // Float twt;
					    //twt=*(imgWts_ptr+ichan+irow*nDataChan);
					    //twt=(imgWts_Chan);
					    vis=hipCmulf(*((hipComplex *)visCube_ptr+ iCiP_offset),phasor);
					    
					    nvalue.x= imgWts_Chan * vis.x;
					    nvalue.y= imgWts_Chan * vis.y;
					  }
					
  					norm.x = norm.y = 0.0;
  					Bool foundCFPeak=False;
  					uInt nMueller=1; //vbs->cfBSt_p.nMueller
  					for (uInt mRow=0;mRow<nMueller; mRow++) 
  					  {
  					    const hipComplex* convFuncV;
  					    Int muellerElementsIndex[4][1] ={{0},{},{},{1}};
  					    Int conjMuellerElementsIndex[4][1] ={{1},{},{},{0}};
					    Int polNdx;
					    if (dataWVal > 0.0) polNdx=muellerElementsIndex[ipol][mRow];
					    else                polNdx=conjMuellerElementsIndex[ipol][mRow];
					    convFuncV = (hipComplex *)cfV[polNdx];
					    
  					    // convOrigin[0]=cfShape[0]/2;	    convOrigin[1]=cfShape[1]/2;
  					    // convOrigin[2]=cfShape[2]/2;	    convOrigin[3]=cfShape[3]/2;
  					    Bool psfOnly=((dopsf==True) && (accumCFs==False));
					    
  					    Int cachedPhaseGradNX=0,cachedPhaseGradNY=0;
  					    hipComplex *cached_PhaseGrad_ptr=NULL;
					    
					    hipComplex tmpNorm; tmpNorm.x=tmpNorm.y=0.0;

  					    tmpNorm = cuaccumulateOnGrid(gridStore, gridInc_l, cached_PhaseGrad_ptr, 
					    				 cachedPhaseGradNX, cachedPhaseGradNY,
					    				 convFuncV, cfInc_l, nvalue,dataWVal,
					    				 iblc, itrc, support, sampling, off, 
					    				 convOrigin, cfShape, loc, igrdpos,
					    				 finitePointingOffsets, psfOnly, foundCFPeak,
					    				 gridHits[XThGrid + YThGrid*subGridShape[0]]);
					    norm.x += tmpNorm.x; norm.y += tmpNorm.y;
  					  }
					
  					//*(sumWt_ptr+targetIMPol+targetIMChan*nGridPol)+= *(imgWts_ptr+ichan+irow*nDataChan)*hipCabsf(norm);
					*(sumWt_ptr+targetIMPol+targetIMChan*nGridPol)+= (imgWts_Chan)*hipCabsf(norm);
  				      }
 				  }
  			      } // End poln-loop
  			  }
  		      }
  		  }
  	      } // End chan-loop
  	  }
      } // End row-loop
    //exit(0);
  }
  //
  //---------------------------------------------------------------------------------
  //
  template <class T>
  __device__
  hipComplex cuaccumulateOnGrid(T* gridStore, const Int* gridInc_p, const hipComplex *cached_phaseGrad_p,
			       const Int cachedPhaseGradNX, const Int cachedPhaseGradNY,
			       const hipComplex* convFuncV, const Int *cfInc_p, hipComplex nvalue,
			       Double wVal, 
			       Int *supBLC_ptr, Int *supTRC_ptr,
			       const Int *support_ptr,
			       Float* scaledSampling_ptr, 
			       Double* off_ptr, Int* convOrigin_ptr, 
			       Int* cfShape, Int* loc_ptr, Int* iGrdpos_ptr,
			       Bool finitePointingOffset,
			       Bool doPSFOnly, Bool& foundCFPeak,
			       Int& gridHits)
  {
    Int iloc_ptr[4]={0,0,0,0};
    // !!! Converting sampling and offset to Ints. Check if this still gives correct results.
    Int scaledSampling_l[2]={(Int)scaledSampling_ptr[0], (Int)scaledSampling_ptr[1]};
    Int off_l[2]={(Int)off_ptr[0], off_ptr[1]};
    
    hipComplex wt;
    //hipComplex cfArea;cfArea.x=1.0; 
    hipComplex norm;norm.x=norm.y=0.0;

    //    Bool finitePointingOffset_l=finitePointingOffset;
    //    Bool doPSFOnly_l=doPSFOnly;
    Double wVal_l=wVal;
    hipComplex nvalue_l=nvalue;
    
    // Int phaseGradOrigin_l[2]; 

    // phaseGradOrigin_l[0] = cachedPhaseGradNX/2;
    // phaseGradOrigin_l[1] = cachedPhaseGradNY/2;
    
    Int xOff=off_l[0]-1+convOrigin_ptr[0], 
      yOff = off_l[1]-1+convOrigin_ptr[1];

    for(Int iy=supBLC_ptr[1]; iy <= supTRC_ptr[1]; iy++) 
      {
	//iloc_ptr[1]=((scaledSampling_l[1]*iy+off_l[1])-1)+convOrigin_ptr[1];
	iloc_ptr[1]=scaledSampling_l[1]*iy+yOff;
	iGrdpos_ptr[1]=loc_ptr[1]+iy;

	for(Int ix=supBLC_ptr[0]; ix <= supTRC_ptr[0]; ix++) 
	  {
	    //iloc_ptr[0]=((scaledSampling_l[0]*ix+off_l[0])-1)+convOrigin_ptr[0];
	    iloc_ptr[0]=scaledSampling_l[0]*ix+xOff;
	    iGrdpos_ptr[0]=loc_ptr[0]+ix;
	    {
	      if (ix==0 and iy==0) foundCFPeak=True;

	      if (ix+support_ptr[0]==threadIdx.x and iy+support_ptr[1]==threadIdx.y)
		{
		  // printf("       # : %d %d %d %d \n",ix+support_ptr[0], iy+support_ptr[1], threadIdx.x,threadIdx.y);
		  wt = CU_GET_FROM_4DARRAY(convFuncV, iloc_ptr,cfInc_p);///cfArea;

		  // !!!UNCOMMENT THE FOLLOWING 2 LINES
		  // if (wVal > 0.0) {wt = hipConjf(wt);}
		  norm = hipCaddf(norm,wt);
	      

		  // !!! ENABLE COMPUTING
		  // The following uses raw index on the 4D grid
		  cuaddTo4DArray(gridStore,iGrdpos_ptr,gridInc_p, nvalue,wt);
		  //gridHits++;
		}
	    }
	  }
      }
    return norm;
  }
  //
  //---------------------------------------------------------------------------------
  //
  __device__
  void cusgrid(Double pos[3], Int loc[3], Double off[3], hipComplex* phasor, 
	      const Int irow, const Double* uvw_ptr, const Double dphase, 
	      const Double freq, const Double* scale, const Double* offset,
	      const Float sampling[2])
  {
    Float phase;
    Float uvw_l[3]={0.0,0.0,0.0}; // This allows gridding of weights
    Float LambdaInv=freq/299792458.0;
    Float offset_l[3]={offset[0],offset[1],offset[2]};
    Float freq_l=freq;
    Float scale_l[3]={scale[0],scale[1],scale[2]};
    Float pos_l[3],sampling_l[3]={sampling[0], sampling[1], sampling[2]};
    // centered on the uv-origin
    //  if (uvw.nelements() > 0) for(Int i=0;i<3;i++) uvw_l[i]=uvw(i,irow);
    // if (uvw.nelements() > 0) for(Int i=0;i<3;i++) uvw_l[i]=uvw_ptr[i+irow*3];
    if (uvw_ptr != NULL) 
      {
	for(Int i=0;i<3;i++) 
	  {
	    uvw_l[i]=uvw_ptr[i+irow*3];
	  }
      }
    // else 
    //   printf("cusgrid::UVW == 0\n");
    
    pos_l[2]=0;//__fsqrt_rn(abs(scale_l[2]*uvw_l[2]*LambdaInv))+offset_l[2];
    loc[2]=0;//NINT(pos_l[2]);
    off[2]=0;
    
    for(Int idim=0;idim<2;idim++)
      {
	pos_l[idim]=scale_l[idim]*uvw_l[idim]*LambdaInv+(offset_l[idim]);
	loc[idim]=NINT(pos_l[idim]);
	//	off[idim]=SynthesisUtils::nint((loc[idim]-pos_l[idim])*sampling[idim]+1);
	off[idim]=NINT((loc[idim]-(Float)pos_l[idim])*sampling[idim]);
      }
    
    if (fabs(dphase) >= 1e-8)
      {
	phase=-2.0*M_PI*dphase*LambdaInv;
	Float sp,cp;
	sincos(phase,&sp,&cp);
	(*phasor).x=cp;
	(*phasor).y=sp;
      }
    else
      {
	(*phasor).x=1.0;
	(*phasor).y=0.0;
      }
    pos[0]=pos_l[0]; pos[1]=pos_l[1]; pos[2]=pos_l[2];
  }
  //
  //---------------------------------------------------------------------------------
  //
  __device__
  void cuaddTo4DArray(Complex *store, const Int *iPos, const Int* inc, 
		      hipComplex nvalue, hipComplex wt)
  {
    hipComplex tmp=hipCmulf(nvalue,wt);

    int n=iPos[0] + iPos[1]*inc[1] + iPos[2]*inc[2] +iPos[3]*inc[3];
    ((hipComplex *)store)[n].x += tmp.x;
    ((hipComplex *)store)[n].y += tmp.y;


  }
  //
  //---------------------------------------------------------------------------------
  //
  __device__
  void cuaddTo4DArray(DComplex *store, const Int *iPos, const Int* inc, 
		      hipComplex nvalue, hipComplex wt)
  {
    hipComplex tmp=hipCmulf(nvalue,wt);
    int n=iPos[0] + iPos[1]*inc[1] + iPos[2]*inc[2] +iPos[3]*inc[3];

    ((hipDoubleComplex *)store)[n].x += tmp.x;
    ((hipDoubleComplex *)store)[n].y += tmp.y;

    // hipComplex tmp;
    // tmp.x=((hipDoubleComplex *)store)[iPos[0]].x;
    // tmp.y=((hipDoubleComplex *)store)[iPos[0]].y;
  }
  //
  //---------------------------------------------------------------------------------
  //
  __device__
  void cucalcIntersection(const Int blc1[2], const Int trc1[2], 
			  const Float blc2[2], const Float trc2[2],
			  Float blc[2], Float trc[2])
  {
    Float dblc, dtrc;
    for (Int i=0;i<2;i++)
      {
        dblc = blc2[i] - blc1[i];
        dtrc = trc2[i] - trc1[i];

        if ((dblc >= 0) and (dtrc >= 0))
	  {
            blc[i] = blc1[i] + dblc;
            trc[i] = trc2[i] - dtrc;
	  }
        else if ((dblc >= 0) and (dtrc < 0))
	  {
            blc[i] = blc1[i] + dblc;
            trc[i] = trc1[i] + dtrc;
	  }
        else if ((dblc < 0) and (dtrc >= 0))
	  {
            blc[i] = blc2[i] - dblc;
            trc[i] = trc2[i] - dtrc;
	  }
        else
	  {
            blc[i] = blc2[i] - dblc;
            trc[i] = trc1[i] + dtrc;
	  }
      }
  }
  //
  // Check if the two rectangles interset (courtesy U.Rau).
  //
  __device__
  Bool cucheckIntersection(const Int blc1[2], const Int trc1[2], const Float blc2[2], const Float trc2[2])
  {
    // blc1[2] = {xmin1, ymin1}; 
    // blc2[2] = {xmin2, ymin2};
    // trc1[2] = {xmax1, ymax1};
    // trc2[2] = {xmax2, ymax2};

    if ((blc1[0] > trc2[0]) || (trc1[0] < blc2[0]) || (blc1[1] > trc2[1]) || (trc1[1] < blc2[1])) 
      return False;
    else
      return True;
  }
  //
  //---------------------------------------------------------------------------------
  //
  __device__
  Bool cucomputeSupport(const uInt *BLCXi_ptr, const uInt *BLCYi_ptr,
		       const uInt *TRCXi_ptr, const uInt *TRCYi_ptr,
		       const uInt subGridShape[2],
		       const Int XThGrid, const Int YThGrid,
		       const Int support[2], const Float sampling[2],
		       const Double pos[2], const Int loc[3],
		       Float iblc[2], Float itrc[2])
  {

    Int sup[2] = {support[0], support[1]};
    
    Int blc[2] = {BLCXi_ptr[XThGrid + YThGrid*subGridShape[0]], BLCYi_ptr[XThGrid + YThGrid*subGridShape[0]]};
    Int trc[2] = {TRCXi_ptr[XThGrid + YThGrid*subGridShape[0]], TRCYi_ptr[XThGrid + YThGrid*subGridShape[0]]};
    
    Float vblc[2]={pos[0]-sup[0],pos[1]-sup[1]}, vtrc[2]={pos[0]+sup[0],pos[1]+sup[1]};
    
    if (cucheckIntersection(blc,trc,vblc,vtrc))
      {
	cucalcIntersection(blc,trc,vblc,vtrc,iblc,itrc);
	return True;
      }
    return False;
  }
#include "cDataToGridImpl_def.h"
  
};
